#include "hip/hip_runtime.h"
#include "distconv/util/nvshmem.hpp"
#include "distconv/util/util_mpi.hpp"
#include "distconv/util/util_cuda.hpp"

namespace distconv {
namespace util {
namespace nvshmem {

#ifdef DISTCONV_HAS_NVSHMEM
void initialize(MPI_Comm comm) {
  util::MPIRootPrintStreamInfo() << "Initializing NVSHMEM with MPI";
  nvshmemx_init_attr_t attr;
  attr.mpi_comm = &comm;
  DISTCONV_CHECK_NVSHMEM(nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr));
}

void finalize() {
  util::MPIRootPrintStreamInfo() << "Finalizing NVSHMEM";
  nvshmem_finalize();
}

void barrier() {
  nvshmem_barrier_all();
}

namespace internal {

__global__ void sync_pairwise_kernel(int peer,
                                     bool notify, bool wait,
                                     SyncType sync_type,
                                     PairwiseSyncDevice sync) {
  sync.sync(peer, notify, wait, sync_type);
}

__global__ void notify_kernel(int peer,
                              SyncType sync_type,
                              PairwiseSyncDevice sync) {
  sync.notify(peer, sync_type);
}

__global__ void wait_kernel(PairwiseSyncDevice sync) {
  sync.wait();
}

__global__ void inc_counter_kernel(PairwiseSyncDevice sync) {
  sync.inc_counter();
}

} // namespace internal

void PairwiseSync::sync(int peer, bool notify, bool wait,
                        SyncType sync_type, hipStream_t stream) {
  if (peer == MPI_PROC_NULL) return;
  internal::sync_pairwise_kernel<<<1, 1, 0, stream>>>(
      peer, notify, wait, sync_type, get_for_device());
}

void PairwiseSync::notify(int peer, SyncType sync_type,
                          hipStream_t stream) {
  if (peer == MPI_PROC_NULL) return;
  internal::notify_kernel<<<1, 1, 0, stream>>>(
      peer, sync_type, get_for_device());
}

void PairwiseSync::wait(hipStream_t stream) {
  internal::wait_kernel<<<1, 1, 0, stream>>>(get_for_device());
}

void PairwiseSync::inc_counter(hipStream_t stream) {
  internal::inc_counter_kernel<<<1, 1, 0, stream>>>(get_for_device());
}

void PairwiseSync::alloc_buffers() {
  if (m_shmem_counter) {
    // already allocated
    return;
  }
  CounterType *shmem_counter = static_cast<CounterType*>(
      nvshmem_malloc(sizeof(CounterType)));
  //util::MPIPrintStreamDebug() << "shmem flag: " << p;
  if (shmem_counter == nullptr) {
    util::MPIPrintStreamError() << "Allocation of shmem buffer failed";
    throw std::exception();
  }
  DISTCONV_CHECK_CUDA(hipMemset(shmem_counter, 0, sizeof(CounterType)));
  // Make sure the memset is completed
  DISTCONV_CHECK_CUDA(hipStreamSynchronize(0));
  barrier();
  m_shmem_counter = std::shared_ptr<CounterType>(
      shmem_counter, [](CounterType *ptr) { nvshmem_free(ptr); });

  // Setup the device counter variable
  CounterType *local_counter = nullptr;
  DISTCONV_CHECK_CUDA(hipMalloc(&local_counter, sizeof(CounterType)));
  CounterType counter_init = 1;
  DISTCONV_CHECK_CUDA(hipMemcpy(
      local_counter, &counter_init,
      sizeof(CounterType), hipMemcpyHostToDevice));
  m_local_counter = std::shared_ptr<CounterType>(
      local_counter, [](CounterType *ptr) {
                       DISTCONV_CHECK_CUDA(hipFree(ptr)); });
}

PairwiseSyncDevice PairwiseSync::get_for_device() {
  return PairwiseSyncDevice(m_local_counter.get(), m_shmem_counter.get());
}

namespace internal {

__global__ void sync_kernel(int peer, bool notify, bool wait,
                            SyncType sync_type, int idx,
                            SyncArrayDevice sync) {
  sync.sync(peer, notify, wait, sync_type, idx);
}

__global__ void notify_kernel(int peer, SyncType sync_type, int idx,
                              SyncArrayDevice sync) {
  sync.notify(peer, sync_type, idx);
}

__global__ void wait_kernel(int idx, SyncArrayDevice sync) {
  sync.wait(idx);
}

__global__ void inc_counter_kernel(int idx, SyncArrayDevice sync) {
  sync.inc_counter(idx);
}

} // namespace internal

void SyncArray::sync(int peer, bool notify, bool wait,
                     SyncType sync_type, int idx, hipStream_t stream) {
  if (peer == MPI_PROC_NULL) return;
  internal::sync_kernel<<<1, 1, 0, stream>>>(
      peer, notify, wait, sync_type, idx, get_for_device());
}

void SyncArray::notify(int peer, SyncType sync_type, int idx,
                       hipStream_t stream) {
  if (peer == MPI_PROC_NULL) return;
  internal::notify_kernel<<<1, 1, 0, stream>>>(
      peer, sync_type, idx, get_for_device());
}

void SyncArray::wait(int idx, hipStream_t stream) {
  internal::wait_kernel<<<1, 1, 0, stream>>>(idx, get_for_device());
}

void SyncArray::inc_counter(int idx, hipStream_t stream) {
  internal::inc_counter_kernel<<<1, 1, 0, stream>>>(idx, get_for_device());
}

void SyncArray::alloc_counters() {
  if (m_shmem_counter) {
    // already allocated
    return;
  }
  if (m_size == 0) {
    // nothing to allocate
    return;
  }
  CounterType *shmem_counter = static_cast<CounterType*>(
      nvshmem_malloc(sizeof(CounterType) * m_size));
  //util::MPIPrintStreamDebug() << "shmem flag: " << p;
  if (shmem_counter == nullptr) {
    util::MPIPrintStreamError() << "Allocation of shmem buffer failed";
    throw std::exception();
  }
  m_shmem_counter = std::shared_ptr<CounterType>(
      shmem_counter, [](CounterType *ptr) { nvshmem_free(ptr); });
  // Setup the device counter variable
  CounterType *local_counter = static_cast<CounterType*>(
      nvshmem_malloc(sizeof(CounterType) * m_size));
  if (shmem_counter == nullptr) {
    util::MPIPrintStreamError() << "Allocation of local buffer failed";
    throw std::exception();
  }
  m_local_counter = std::shared_ptr<CounterType>(
      local_counter, [](CounterType *ptr) { nvshmem_free(ptr); });
  init_counters();
}

void SyncArray::init_counters() {
  DISTCONV_CHECK_CUDA(hipMemset(
      m_shmem_counter.get(), 0, sizeof(CounterType) * m_size));
  std::vector<CounterType> init(m_size,  1);
  DISTCONV_CHECK_CUDA(hipMemcpy(
      m_local_counter.get(), init.data(),
      sizeof(CounterType) * m_size, hipMemcpyHostToDevice));
  // Make sure the memset is completed
  DISTCONV_CHECK_CUDA(hipStreamSynchronize(0));
  barrier();
}

void SyncArray::ensure_size(size_t size) {
  if (m_size < size) {
    m_size = size;
    alloc_counters();
  }
}

SyncArrayDevice SyncArray::get_for_device() {
  return SyncArrayDevice(m_local_counter.get(), m_shmem_counter.get());
}

#endif // DISTCONV_HAS_NVSHMEM

} // namespace nvshmem
} // namespace util
} // namespace distconv
