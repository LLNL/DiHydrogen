#include "hip/hip_runtime.h"
#include "distconv/cudnn/cross_entropy.hpp"
#include "distconv/util/util_mpi.hpp"
#include "distconv/util/util_cuda.hpp"
#include "distconv/tensor/algorithms_cuda.hpp"

#include <limits>

#include <cub/block/block_reduce.cuh>

using distconv::tensor::LocaleMPI;
using distconv::tensor::CUDAAllocator;

template <typename DataType>
using TensorCUDA = distconv::tensor::Tensor<DataType, LocaleMPI, CUDAAllocator>;
using CrossEntopyCUDNN = distconv::CrossEntropy<distconv::cudnn::BackendCUDNN>;

namespace distconv {
namespace cross_entropy {

/*
  - gridDim.y == number of samples
  - Each sample is taken care by gridDim.x blocks
 */
template <typename DataType, int BLOCK_SIZE>
__global__ void fp_local(const DataType * __restrict__ prediction,
                         const DataType * __restrict__ ground_truth,
                         DataType * __restrict__ y,
                         index_t sample_size,
                         int thread_work_size) {
  const int tid = threadIdx.x;
  const int sample_idx = blockIdx.y;

  prediction += sample_idx * sample_size;
  ground_truth += sample_idx * sample_size;

  index_t offset = tid + blockIdx.x * BLOCK_SIZE;
  const int offset_stride = BLOCK_SIZE * gridDim.x;
  const index_t offset_limit = min(
      sample_size, offset + offset_stride * thread_work_size);

  auto psum = DataType(0.);
  for (; offset < offset_limit; offset += offset_stride) {
    const auto xhat = ground_truth[offset];
    if (xhat > DataType(0.)) {
      const auto x = prediction[offset];
      psum += - xhat * log(x);
    }
  }

  using BlockReduce = hipcub::BlockReduce<DataType, BLOCK_SIZE>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  psum = BlockReduce(temp_storage).Sum(psum);

  if (tid == 0) {
    atomicAdd(&y[sample_idx], psum);
  }
}


/*
  - gridDim.y == number of samples
  - Each sample is taken care by gridDim.x blocks
 */
template <typename DataType, int BLOCK_SIZE>
__global__ void bp_local(const DataType * __restrict__ x_pred,
                         const DataType * __restrict__ x_truth,
                         const DataType * __restrict__ dy,
                         DataType * __restrict__ dx_pred,
                         DataType * __restrict__ dx_truth,
                         index_t sample_size,
                         int thread_work_size) {
  const int tid = threadIdx.x;
  const int sample_idx = blockIdx.y;

  x_pred += sample_idx * sample_size;
  dx_pred += sample_idx * sample_size;
  x_truth += sample_idx * sample_size;
  dx_truth += sample_idx * sample_size;

  index_t offset = tid + blockIdx.x * BLOCK_SIZE;
  const int offset_stride = BLOCK_SIZE * gridDim.x;
  const index_t offset_limit = min(
      sample_size, offset + offset_stride * thread_work_size);

  const auto dy_sample = dy[sample_idx];
  for (; offset < offset_limit; offset += offset_stride) {
    const auto x = x_pred[offset];
    const auto xhat = x_truth[offset];
    dx_pred[offset] = (xhat > DataType(0.)) ?
        - dy_sample * xhat / x : DataType(0.);
    dx_truth[offset] = - dy_sample * log(x);
  }
}

} // namespace cross_entropy

template <typename Tensor>
int CrossEntopyCUDNN::forward(const Tensor &x_pred, const Tensor &x_truth,
                              Tensor &y) {
  using DataType = typename Tensor::data_type;
  util::MPIPrintStreamDebug()
      << "Cross entropy FP: " << x_pred << ", "
      << x_truth << ", " << y;

  constexpr int block_size = 256;
  constexpr int thread_work_size = 8;

  // Assumes no halo for simplicity
  assert_eq(x_pred.get_local_size(), x_pred.get_local_real_size());
  assert_eq(x_truth.get_local_size(), x_truth.get_local_real_size());

  const auto num_samples = x_pred.get_local_shape()[-1];

  if (num_samples == 0) return 0;

  y.zero(m_be.get_stream());

  if (x_pred.get_local_size() > 0) {
    auto sample_size = x_pred.get_local_size() / num_samples;
    auto num_blocks_per_sample = util::ceil(
        sample_size, (index_t)block_size * thread_work_size);

    dim3 bdim(block_size);
    dim3 gdim(num_blocks_per_sample, num_samples);

    cross_entropy::fp_local<DataType, block_size>
        <<<gdim, bdim, 0, m_be.get_stream()>>>(
            x_pred.get_const_buffer(), x_truth.get_const_buffer(),
            y.get_buffer(), sample_size, thread_work_size);
  }

  if (m_num_procs_per_sample > 1) {
    Al::Allreduce<Al::NCCLBackend, DataType>(
        y.get_buffer(), num_samples,
        Al::ReductionOperator::sum, *m_al.get());
  }

  return 0;
}

template <typename Tensor>
int CrossEntopyCUDNN::backward(const Tensor &x_pred, const Tensor &x_truth,
                               const Tensor &dy, Tensor &dx_pred,
                               Tensor &dx_truth) {
  using DataType = typename Tensor::data_type;
  util::MPIPrintStreamDebug()
      << "Cross entropy BP: " << dy << ", " << dx_pred << ", " << dx_truth;

  constexpr int block_size = 256;
  constexpr int thread_work_size = 8;

  // Assumes no halo for simplicity
  assert_eq(dx_pred.get_local_size(), dx_pred.get_local_real_size());
  assert_eq(dx_truth.get_local_size(), dx_truth.get_local_real_size());

  if (x_pred.get_local_size() == 0) return 0;

  auto num_samples = x_pred.get_local_shape()[-1];
  auto sample_size = x_pred.get_local_size() / num_samples;
  auto num_blocks_per_sample = util::ceil(
      sample_size, (index_t)block_size * thread_work_size);

  dim3 bdim(block_size);
  dim3 gdim(num_blocks_per_sample, num_samples);

  cross_entropy::bp_local<DataType, block_size>
      <<<gdim, bdim, 0, m_be.get_stream()>>>(
          x_pred.get_const_buffer(), x_truth.get_const_buffer(),
          dy.get_const_buffer(),
          dx_pred.get_buffer(), dx_truth.get_buffer(),
          sample_size, thread_work_size);
  return 0;
}

template
int CrossEntopyCUDNN::forward<TensorCUDA<float>>(
    const TensorCUDA<float> &x_pred, const TensorCUDA<float> &x_truth,
    TensorCUDA<float> &y);

template
int CrossEntopyCUDNN::backward<TensorCUDA<float>>(
    const TensorCUDA<float> &x_pred, const TensorCUDA<float> &x_truth,
    const TensorCUDA<float> &dy, TensorCUDA<float> &dx_pred,
    TensorCUDA<float> &dx_truth);

} // namespace distconv
