#include "hip/hip_runtime.h"
#include "distconv/dnn_backend/cross_entropy.hpp"
#include "distconv/runtime_gpu.hpp"
#include "distconv/tensor/algorithms_cuda.hpp"
#include "distconv/tensor/tensor_mpi.hpp"
#include "distconv/util/util_gpu.hpp"
#include "distconv/util/util_mpi.hpp"

#include <limits>

#if H2_HAS_CUDA
#include <cub/block/block_reduce.cuh>
namespace cubns = cub;
#elif H2_HAS_ROCM
#include <hipcub/block/block_reduce.hpp>
namespace cubns = hipcub;
#endif

using distconv::tensor::CUDAAllocator;
using distconv::tensor::LocaleMPI;

template <typename DataType>
using TensorCUDA = distconv::tensor::Tensor<DataType, LocaleMPI, CUDAAllocator>;

namespace distconv
{
namespace
{

/*
  - gridDim.y == number of samples
  - Each sample is taken care by gridDim.x blocks
 */
template <typename DataType, int BLOCK_SIZE>
__global__ void fp_local(DataType const* __restrict__ prediction,
                         DataType const* __restrict__ ground_truth,
                         DataType* __restrict__ y,
                         index_t const sample_size,
                         index_t const sample_spatial_size,
                         index_t const sample_channel_size,
                         bool const use_labels,
                         int thread_work_size)
{
  int const tid = threadIdx.x;
  int const sample_idx = blockIdx.y;

  prediction += sample_idx * sample_size;
  ground_truth += sample_idx * sample_size;

  index_t offset = tid + blockIdx.x * BLOCK_SIZE;
  int const offset_stride = BLOCK_SIZE * gridDim.x;
  index_t const offset_limit =
    min(sample_size, offset + offset_stride * thread_work_size);

  auto psum = DataType(0.);
  for (; offset < offset_limit; offset += offset_stride)
  {
    DataType xhat;
    if (use_labels)
    {
      auto const spatial = offset % sample_spatial_size;
      auto const channel = (offset / sample_spatial_size) % sample_channel_size;
      auto const sample = offset / sample_spatial_size / sample_channel_size;
      auto const offset_truth = spatial + sample * sample_spatial_size;
      int const truth_label = ground_truth[offset_truth];
      xhat = DataType(truth_label == channel ? 1. : 0.);
    }
    else
    {
      xhat = ground_truth[offset];
    }
    if (xhat > DataType(0.))
    {
      auto const x = prediction[offset];
      psum += -xhat * log(x);
    }
  }

  using BlockReduce = cubns::BlockReduce<DataType, BLOCK_SIZE>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  psum = BlockReduce(temp_storage).Sum(psum);

  if (tid == 0)
  {
    atomic_add(&y[sample_idx], psum);
  }
}

/*
  - gridDim.y == number of samples
  - Each sample is taken care by gridDim.x blocks
 */
template <typename DataType, int BLOCK_SIZE>
__global__ void bp_local(DataType const* __restrict__ x_pred,
                         DataType const* __restrict__ x_truth,
                         DataType const* __restrict__ dy,
                         DataType* __restrict__ dx_pred,
                         DataType* __restrict__ dx_truth,
                         index_t const sample_size,
                         index_t const sample_spatial_size,
                         index_t const sample_channel_size,
                         bool const use_labels,
                         int thread_work_size)
{
  int const tid = threadIdx.x;
  int const sample_idx = blockIdx.y;

  x_pred += sample_idx * sample_size;
  dx_pred += sample_idx * sample_size;
  x_truth += sample_idx * sample_size;
  dx_truth += sample_idx * sample_size;

  index_t offset = tid + blockIdx.x * BLOCK_SIZE;
  int const offset_stride = BLOCK_SIZE * gridDim.x;
  index_t const offset_limit =
    min(sample_size, offset + offset_stride * thread_work_size);

  auto const dy_sample = dy[sample_idx];
  for (; offset < offset_limit; offset += offset_stride)
  {
    auto const x = x_pred[offset];
    DataType xhat;
    if (use_labels)
    {
      auto const spatial = offset % sample_spatial_size;
      auto const channel = (offset / sample_spatial_size) % sample_channel_size;
      auto const sample = offset / sample_spatial_size / sample_channel_size;
      auto const offset_truth = spatial + sample * sample_spatial_size;
      int const truth_label = x_truth[offset_truth];
      xhat = DataType(truth_label == channel ? 1. : 0.);
    }
    else
    {
      xhat = x_truth[offset];
    }
    dx_pred[offset] =
      (xhat > DataType(0.)) ? -dy_sample * xhat / x : DataType(0.);
    if (!use_labels)
    {
      dx_truth[offset] = -dy_sample * log(x);
    }
  }
}

}  // namespace

template <typename Tensor>
int CrossEntropy<BackendDNNLib>::forward(Tensor const& x_pred,
                                         Tensor const& x_truth,
                                         Tensor& y)
{
  using DataType = typename Tensor::data_type;
  util::MPIPrintStreamDebug()
    << "Cross entropy FP: " << x_pred << ", " << x_truth << ", " << y;

  constexpr int block_size = 256;
  constexpr int thread_work_size = 8;

  // Assumes no halo for simplicity
  assert_eq(x_pred.get_local_size(), x_pred.get_local_real_size());
  assert_eq(x_truth.get_local_size(), x_truth.get_local_real_size());

  auto const num_samples = x_pred.get_local_shape()[-1];

  if (num_samples == 0)
    return 0;

  y.zero(m_stream);

  if (x_pred.get_local_size() > 0)
  {
    auto sample_size = x_pred.get_local_size() / num_samples;
    auto num_blocks_per_sample =
      util::ceil(sample_size, (index_t) block_size * thread_work_size);

    dim3 bdim(block_size);
    dim3 gdim(num_blocks_per_sample, num_samples);

    auto const sample_channel_size =
      x_pred.get_local_shape()[x_pred.get_num_spatial_dims()];
    auto const sample_spatial_size = sample_size / sample_channel_size;
    assert_eq(sample_channel_size * sample_spatial_size, sample_size);

    fp_local<DataType, block_size>
      <<<gdim, bdim, 0, m_stream>>>(x_pred.get_const_buffer(),
                                    x_truth.get_const_buffer(),
                                    y.get_buffer(),
                                    sample_size,
                                    sample_spatial_size,
                                    sample_channel_size,
                                    m_use_labels,
                                    thread_work_size);
  }

  if (m_num_procs_per_sample > 1)
  {
    Al::Allreduce<Al::NCCLBackend, DataType>(
      y.get_buffer(), num_samples, Al::ReductionOperator::sum, *m_al.get());
  }

  return 0;
}

template <typename Tensor>
int CrossEntropy<BackendDNNLib>::backward(Tensor const& x_pred,
                                          Tensor const& x_truth,
                                          Tensor& dy,
                                          Tensor& dx_pred,
                                          Tensor& dx_truth)
{
  using DataType = typename Tensor::data_type;
  util::MPIPrintStreamDebug()
    << "Cross entropy BP: " << dy << ", " << dx_pred << ", " << dx_truth;

  if (m_num_procs_per_sample > 1)
  {
    auto const num_samples = x_pred.get_local_shape()[-1];
    Al::Bcast<Al::NCCLBackend, DataType>(
      dy.get_buffer(), num_samples, 0, *m_al.get());
  }

  constexpr int block_size = 256;
  constexpr int thread_work_size = 8;

  // Assumes no halo for simplicity
  assert_eq(dx_pred.get_local_size(), dx_pred.get_local_real_size());
  assert_eq(dx_truth.get_local_size(), dx_truth.get_local_real_size());

  if (x_pred.get_local_size() == 0)
    return 0;

  auto num_samples = x_pred.get_local_shape()[-1];
  auto sample_size = x_pred.get_local_size() / num_samples;
  auto num_blocks_per_sample =
    util::ceil(sample_size, (index_t) block_size * thread_work_size);

  dim3 bdim(block_size);
  dim3 gdim(num_blocks_per_sample, num_samples);

  auto const sample_channel_size =
    x_pred.get_local_shape()[x_pred.get_num_spatial_dims()];
  auto const sample_spatial_size = sample_size / sample_channel_size;
  assert_eq(sample_channel_size * sample_spatial_size, sample_size);

  bp_local<DataType, block_size>
    <<<gdim, bdim, 0, m_stream>>>(x_pred.get_const_buffer(),
                                  x_truth.get_const_buffer(),
                                  dy.get_const_buffer(),
                                  dx_pred.get_buffer(),
                                  dx_truth.get_buffer(),
                                  sample_size,
                                  sample_spatial_size,
                                  sample_channel_size,
                                  m_use_labels,
                                  thread_work_size);
  return 0;
}

#define PROTO(T)                                                               \
  template int CrossEntropy<BackendDNNLib>::forward<TensorCUDA<T>>(            \
    const TensorCUDA<T>& x_pred,                                               \
    const TensorCUDA<T>& x_truth,                                              \
    TensorCUDA<T>& y);                                                         \
  template int CrossEntropy<BackendDNNLib>::backward<TensorCUDA<T>>(           \
    const TensorCUDA<T>& x_pred,                                               \
    const TensorCUDA<T>& x_truth,                                              \
    TensorCUDA<T>& dy,                                                         \
    TensorCUDA<T>& dx_pred,                                                    \
    TensorCUDA<T>& dx_truth);

PROTO(float)
PROTO(double)
#undef PROTO

}  // namespace distconv
