#include "hip/hip_runtime.h"
#include "distconv/distconv.hpp"
#include "distconv/dnn_backend/batchnorm.hpp"
#include "distconv/runtime_gpu.hpp"
#include "distconv/tensor/algorithms_cuda.hpp"
#include "distconv/util/util_gpu.hpp"
#include "distconv/util/util_mpi.hpp"

#include <type_traits>

#if H2_HAS_CUDA
#include <cub/block/block_reduce.cuh>
namespace cubns = cub;
#elif H2_HAS_ROCM
#include <hipcub/block/block_reduce.hpp>
namespace cubns = hipcub;
#endif

using distconv::index_t;
using distconv::tensor::CUDAAllocator;
using distconv::tensor::LocaleMPI;
#ifdef DISTCONV_HAS_NVSHMEM
using distconv::tensor::AllreduceNVSHMEM;
using distconv::tensor::AllreduceNVSHMEMDevice;
#endif

using distconv::get_channel_dim;
using distconv::get_sample_dim;

template <typename DataType>
using Tensor = distconv::tensor::Tensor<DataType, LocaleMPI, CUDAAllocator>;

namespace distconv
{
namespace batchnorm
{
namespace
{

template <int ND, typename DataType, int BLOCK_SIZE>
__global__ void
channel_sums_and_sqsums_kernel(DataType const* __restrict__ input,
                               DataType* __restrict__ sums,
                               DataType* __restrict__ sqsums,
                               tensor::Array<ND> shape,
                               tensor::Array<ND> input_strides)
{
  int const tid = threadIdx.x;
  index_t const gidx = threadIdx.x + blockIdx.x * blockDim.x;
  int const ch_idx = blockIdx.y;
  int const num_channels = shape[get_channel_dim()];
  int const num_samples = shape[get_sample_dim()];

  DataType sum = DataType(0);
  DataType sqsum = DataType(0);

  index_t const channel_size = shape.get_size() / num_channels / num_samples;

  if (gidx < channel_size)
  {
    index_t offset = gidx;
    index_t input_offset = 0;
    for (int d = 0; d < ND - 2; ++d)
    {
      int idx = offset % shape[d];
      input_offset += idx * input_strides[d];
      offset /= shape[d];
    }
    input_offset += ch_idx * input_strides[-2];
    for (int s = 0; s < num_samples; ++s)
    {
      DataType const x = input[input_offset];
      sum += x;
      sqsum += x * x;

      input_offset += input_strides[-1];
    }
  }

  using BlockReduce = cubns::BlockReduce<DataType, BLOCK_SIZE>;
  __shared__ typename BlockReduce::TempStorage temp_storage_sum;
  __shared__ typename BlockReduce::TempStorage temp_storage_sqsum;
  sum = BlockReduce(temp_storage_sum).Sum(sum);
  sqsum = BlockReduce(temp_storage_sqsum).Sum(sqsum);
  // Output channel sum to global memory
  if (tid == 0)
  {
    atomic_add(&sums[ch_idx], sum);
    atomic_add(&sqsums[ch_idx], sqsum);
  }
}

template <int ND, typename DataType, int BLOCK_SIZE, typename DataTypeV>
__global__ void
channel_sums_and_sqsums_opt_kernel(DataTypeV const* __restrict__ input,
                                   DataType* __restrict__ sums,
                                   DataType* __restrict__ sqsums,
                                   int const num_channels,
                                   int const num_samples,
                                   index_t const spatial_size,
                                   index_t const spatial_real_size)
{
  int const tid = threadIdx.x;
  int const idx = threadIdx.x + blockIdx.x * blockDim.x;
  int const ch_idx = blockIdx.y;
  auto const sample_offset = spatial_real_size * num_channels;

  auto sum = DataType(0);
  auto sqsum = DataType(0);
  index_t offset = spatial_real_size * ch_idx;
  for (int s = 0; s < num_samples; ++s)
  {
    for (int i = idx; i < spatial_size; i += BLOCK_SIZE * gridDim.x)
    {
      auto const x = input[offset + i];
      sum += util::sum(x);
      sqsum += util::sum(x * x);
    }
    offset += sample_offset;
  }

  using BlockReduce = cubns::BlockReduce<DataType, BLOCK_SIZE>;
  __shared__ typename BlockReduce::TempStorage temp_storage_sum;
  __shared__ typename BlockReduce::TempStorage temp_storage_sqsum;
  sum = BlockReduce(temp_storage_sum).Sum(sum);
  sqsum = BlockReduce(temp_storage_sqsum).Sum(sqsum);
  // Output channel sum to global memory
  if (tid == 0)
  {
    atomic_add(&sums[ch_idx], sum);
    atomic_add(&sqsums[ch_idx], sqsum);
  }
}

template <int ND, typename Tensor>
void channel_sums_and_sqsums_opt(int num_samples,
                                 Tensor const& input,
                                 Tensor& sums,
                                 Tensor& sqsums,
                                 h2::gpu::DeviceStream stream)
{
  using DataType = typename Tensor::data_type;

  // Do not contribute to the accumulation if the local tensor is not
  // a split root.
  if (input.get_local_size() == 0 || !input.is_split_root())
    return;

  int const num_channels = input.get_local_shape()[get_channel_dim()];
  constexpr int block_size = 256;
  dim3 block_dim(block_size);
  constexpr index_t thread_work_size = 8;
  constexpr auto block_work_size = block_size * thread_work_size;
  index_t spatial_size = input.get_local_size() / num_channels / num_samples;
  index_t spatial_real_size =
    input.get_local_real_size() / num_channels / num_samples;
  // halo size must be also divisible by a vector width for an
  // alignment requirement
  if (spatial_size % 4 == 0
      && ((spatial_real_size - spatial_size) / 2) % 4 == 0)
  {
    using DataTypeV = typename util::GetVectorType<DataType, 4>::type;
    spatial_size /= 4;
    spatial_real_size /= 4;
    auto num_blocks_per_channel = util::ceil(spatial_size, block_work_size);
    dim3 grid_dim(num_blocks_per_channel, num_channels);
    channel_sums_and_sqsums_opt_kernel<ND, DataType, block_size, DataTypeV>
      <<<grid_dim, block_dim, 0, stream>>>(
        reinterpret_cast<DataTypeV const*>(input.get_const_base_ptr()),
        sums.get_base_ptr(),
        sqsums.get_base_ptr(),
        num_channels,
        num_samples,
        spatial_size,
        spatial_real_size);
  }
  else
  {
    using DataTypeV = DataType;
    auto num_blocks_per_channel = util::ceil(spatial_size, block_work_size);
    dim3 grid_dim(num_blocks_per_channel, num_channels);
    channel_sums_and_sqsums_opt_kernel<ND, DataType, block_size, DataType>
      <<<grid_dim, block_dim, 0, stream>>>(input.get_const_base_ptr(),
                                           sums.get_base_ptr(),
                                           sqsums.get_base_ptr(),
                                           num_channels,
                                           num_samples,
                                           spatial_size,
                                           spatial_real_size);
  }
}

template <int ND, typename Tensor>
void channel_sums_and_sqsums(int num_samples,
                             Tensor const& input,
                             Tensor& sums,
                             Tensor& sqsums,
                             h2::gpu::DeviceStream stream)
{
  using DataType = typename Tensor::data_type;
  // Clear GPU memory
  h2::gpu::mem_zero(sums.get_buffer(), sums.get_local_pitched_size(), stream);
  h2::gpu::mem_zero(
    sqsums.get_buffer(), sqsums.get_local_pitched_size(), stream);

  // Do not contribute to the accumulation if the local tensor is not
  // a split root.
  if (input.get_local_size() == 0 || !input.is_split_root())
    return;

  auto overlap = input.get_overlap();
  bool opt_eligible = true;
  for (int i = 0; i < ND - 3; ++i)
  {
    if (overlap[i] != 0)
    {
      opt_eligible = false;
      break;
    }
  }
  if (std::getenv("DISTCONV_DISABLE_BN_OPT"))
  {
    util::MPIRootPrintStreamInfo() << "Disable BN optimization";
    opt_eligible = false;
  }
  if (opt_eligible)
  {
    channel_sums_and_sqsums_opt<ND, Tensor>(
      num_samples, input, sums, sqsums, stream);
    return;
  }

  int const num_channels = input.get_local_shape()[get_channel_dim()];
  constexpr int block_size = 256;
  dim3 block_dim(block_size);
  index_t channel_size = input.get_local_size() / num_channels / num_samples;
  dim3 grid_dim((channel_size + block_size - 1) / block_size, num_channels);
  auto input_strides = input.get_strides();
  auto shape = input.get_local_shape();
  shape[get_sample_dim()] = num_samples;
  // CUDA grid dimension limitation
  assert_always(num_channels < 65535);

  channel_sums_and_sqsums_kernel<ND, DataType, block_size>
    <<<grid_dim, block_dim, 0, stream>>>(input.get_const_base_ptr(),
                                         sums.get_base_ptr(),
                                         sqsums.get_base_ptr(),
                                         shape,
                                         input_strides);
}
}  // namespace

template <typename Tensor>
void channel_sums_and_sqsums(int num_dims,
                             int num_samples,
                             Tensor const& input,
                             Tensor& sums,
                             Tensor& sqsums,
                             h2::gpu::DeviceStream stream)
{
  switch (num_dims)
  {
  case 4:
    channel_sums_and_sqsums<4, Tensor>(
      num_samples, input, sums, sqsums, stream);
    break;
  case 5:
    channel_sums_and_sqsums<5, Tensor>(
      num_samples, input, sums, sqsums, stream);
    break;
  }
}

#define INSTANTIATE_CHANNEL_SUMS_AND_SQSUMS(TYPE)                              \
  template void channel_sums_and_sqsums<Tensor<TYPE>>(                         \
    int num_dims,                                                              \
    int num_samples,                                                           \
    const Tensor<TYPE>& input,                                                 \
    Tensor<TYPE>& sums,                                                        \
    Tensor<TYPE>& sqsums,                                                      \
    h2::gpu::DeviceStream stream);
INSTANTIATE_CHANNEL_SUMS_AND_SQSUMS(float)
INSTANTIATE_CHANNEL_SUMS_AND_SQSUMS(double)
#undef INSTANTIATE_CHANNEL_SUMS_AND_SQSUMS

namespace
{

template <typename DataType>
struct sums_to_statistics_functor
{
  index_t m_num_per_sum;
  DataType m_decay;
  sums_to_statistics_functor(index_t num_per_sum, DataType decay)
    : m_num_per_sum(num_per_sum), m_decay(decay)
  {}

  __device__ void operator()(DataType& global_mean,
                             DataType& global_var,
                             DataType& running_mean,
                             DataType& running_var)
  {
    DataType const mean = global_mean / m_num_per_sum;
    DataType const sqmean = global_var / m_num_per_sum;
    DataType var = sqmean - mean * mean;
    var = var > DataType(0) ? var : DataType(0);
    var *= m_num_per_sum / (m_num_per_sum - DataType(1));
    global_mean = mean;
    global_var = var;

    running_mean = m_decay * running_mean + (DataType(1) - m_decay) * mean;
    running_var = m_decay * running_var + (DataType(1) - m_decay) * var;
  }
};

}  // namespace

template <typename TensorType>
void sums_to_statistics(index_t num_per_sum,
                        typename TensorType::data_type decay,
                        TensorType& global_mean,
                        TensorType& global_var,
                        TensorType& running_mean,
                        TensorType& running_var,
                        h2::gpu::DeviceStream stream)
{
  using DataType = typename TensorType::data_type;
  if (num_per_sum > 0)
  {
    tensor::Transform(global_mean,
                      global_var,
                      running_mean,
                      running_var,
                      sums_to_statistics_functor<DataType>(num_per_sum, decay),
                      stream);
  }
  else
  {
    // Fill global_var with 1. Do the same thing as the corresponding LBANN
    // code.
    tensor::Transform(
      global_var,
      [] __device__(DataType & global_var) { global_var = DataType(1); },
      stream);
  }
}

#define INSTANTIATE_SUMS_TO_STATISTICS(TYPE)                                   \
  template void sums_to_statistics<Tensor<TYPE>>(                              \
    index_t num_per_sum,                                                       \
    TYPE decay,                                                                \
    Tensor<TYPE> & global_mean,                                                \
    Tensor<TYPE> & global_var,                                                 \
    Tensor<TYPE> & running_mean,                                               \
    Tensor<TYPE> & running_var,                                                \
    h2::gpu::DeviceStream stream);
INSTANTIATE_SUMS_TO_STATISTICS(float)
INSTANTIATE_SUMS_TO_STATISTICS(double)
#undef INSTANTIATE_SUMS_TO_STATISTICS

namespace
{

__device__ inline float rsqrt(float x)
{
  return rsqrtf(x);
}

template <int ND, typename DataType>
__global__ void
batch_normalization_kernel(DataType const* __restrict__ input,
                           DataType const* __restrict__ global_mean,
                           DataType const* __restrict__ global_var,
                           DataType const* __restrict__ global_scale,
                           DataType const* __restrict__ global_bias,
                           DataType* __restrict__ output,
                           DataType epsilon,
                           tensor::Array<ND> shape,
                           tensor::Array<ND> input_strides,
                           tensor::Array<ND> output_strides)
{
  int const ch_idx = blockIdx.y;
  int const num_channels = shape[get_channel_dim()];
  int const num_samples = shape[get_sample_dim()];
  DataType const mean = global_mean[ch_idx];
  DataType const var = global_var[ch_idx];
  DataType const scale = global_scale[ch_idx];
  DataType const bias = global_bias[ch_idx];
  DataType const inv_stdev = rsqrt(var + epsilon);

  index_t const gidx = threadIdx.x + blockIdx.x * blockDim.x;
  index_t const channel_size = shape.get_size() / num_channels / num_samples;

  if (gidx < channel_size)
  {
    index_t offset = gidx;
    index_t input_offset = 0, output_offset = 0;
    for (int d = 0; d < ND - 2; ++d)
    {
      int idx = offset % shape[d];
      input_offset += idx * input_strides[d];
      output_offset += idx * output_strides[d];
      offset /= shape[d];
    }
    input_offset += ch_idx * input_strides[-2];
    output_offset += ch_idx * output_strides[-2];
    for (int s = 0; s < num_samples; ++s)
    {
      DataType const x = input[input_offset];
      DataType xhat = (x - mean) * inv_stdev;
      DataType y = scale * xhat + bias;
      output[output_offset] = y;

      input_offset += input_strides[-1];
      output_offset += output_strides[-1];
    }
  }
}

template <int ND, typename DataType, typename DataTypeV>
__global__ void
batch_normalization_opt_kernel(DataTypeV const* __restrict__ input,
                               DataType const* __restrict__ global_mean,
                               DataType const* __restrict__ global_var,
                               DataType const* __restrict__ global_scale,
                               DataType const* __restrict__ global_bias,
                               DataTypeV* __restrict__ output,
                               DataType epsilon,
                               index_t spatial_size,
                               int num_channels)
{
  auto const ch_idx = blockIdx.y;
  auto const sample_idx = blockIdx.z;
  auto const mean = global_mean[ch_idx];
  auto const var = global_var[ch_idx];
  auto const scale = global_scale[ch_idx];
  auto const bias = global_bias[ch_idx];
  auto const inv_stdev = rsqrt(var + epsilon);

  auto const num_threads_per_channel = blockDim.x * gridDim.x;

  auto block_offset = (ch_idx + sample_idx * num_channels) * spatial_size;
  input += block_offset;
  output += block_offset;

  for (index_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < spatial_size;
       idx += num_threads_per_channel)
  {
    auto x = input[idx];
    auto xhat = (x - mean) * inv_stdev;
    auto y = xhat * scale + bias;
    output[idx] = y;
  }
}

template <int ND, typename TensorType>
void batch_normalization_opt(int num_samples,
                             TensorType const& input,
                             TensorType const& mean,
                             TensorType const& var,
                             TensorType const& scale,
                             TensorType const& bias,
                             TensorType& output,
                             typename TensorType::data_type epsilon,
                             h2::gpu::DeviceStream stream)
{
  using DataType = typename TensorType::data_type;
  // local tensors can be empty
  if (output.get_local_size() == 0)
    return;
  assert_eq(num_samples, (int) input.get_local_shape()[get_sample_dim()]);
  int const num_channels = input.get_local_shape()[get_channel_dim()];
  constexpr int block_size = 256;
  dim3 block_dim(block_size);
  index_t channel_size = input.get_local_size() / num_channels / num_samples;
  constexpr index_t thread_work_size = 8;
  constexpr auto block_work_size = block_size * thread_work_size;
  if (channel_size % 4 == 0)
  {
    channel_size /= 4;
    auto num_blocks_per_channel = util::ceil(channel_size, block_work_size);
    dim3 grid_dim(num_blocks_per_channel, num_channels, num_samples);
    using DataTypeV = typename util::GetVectorType<DataType, 4>::type;
    batch_normalization_opt_kernel<ND, DataType, DataTypeV>
      <<<grid_dim, block_dim, 0, stream>>>(
        reinterpret_cast<DataTypeV const*>(input.get_const_buffer()),
        mean.get_const_base_ptr(),
        var.get_const_base_ptr(),
        scale.get_const_base_ptr(),
        bias.get_const_base_ptr(),
        reinterpret_cast<DataTypeV*>(output.get_buffer()),
        epsilon,
        channel_size,
        num_channels);
  }
  else
  {
    auto num_blocks_per_channel = util::ceil(channel_size, block_work_size);
    dim3 grid_dim(num_blocks_per_channel, num_channels, num_samples);
    batch_normalization_opt_kernel<ND, DataType, DataType>
      <<<grid_dim, block_dim, 0, stream>>>(input.get_const_buffer(),
                                           mean.get_const_base_ptr(),
                                           var.get_const_base_ptr(),
                                           scale.get_const_base_ptr(),
                                           bias.get_const_base_ptr(),
                                           output.get_buffer(),
                                           epsilon,
                                           channel_size,
                                           num_channels);
  }
}

template <int ND, typename TensorType>
void batch_normalization(int num_samples,
                         TensorType const& input,
                         TensorType const& mean,
                         TensorType const& var,
                         TensorType const& scale,
                         TensorType const& bias,
                         TensorType& output,
                         typename TensorType::data_type epsilon,
                         h2::gpu::DeviceStream stream)
{
  using DataType = typename TensorType::data_type;

  if (input.get_local_real_shape() == output.get_local_real_shape())
  {
    if (std::getenv("DISTCONV_DISABLE_BN_OPT"))
    {
      util::MPIRootPrintStreamInfo() << "Disable BN optimization";
    }
    else
    {
      batch_normalization_opt<ND, TensorType>(
        num_samples, input, mean, var, scale, bias, output, epsilon, stream);
      return;
    }
  }

  // local tensors can be empty
  if (output.get_local_size() == 0)
    return;
  assert_eq(num_samples, (int) input.get_local_shape()[get_sample_dim()]);
  int const num_channels = input.get_local_shape()[get_channel_dim()];
  constexpr int block_size = 256;
  dim3 block_dim(block_size);
  index_t channel_size = input.get_local_size() / num_channels / num_samples;
  dim3 grid_dim((channel_size + block_size - 1) / block_size, num_channels);
  tensor::Array<ND> input_strides = input.get_strides();
  tensor::Array<ND> output_strides = output.get_strides();
  // CUDA grid dimension limitation
  assert_always(num_channels < 65535);
  tensor::Array<ND> shape = input.get_local_shape();
  batch_normalization_kernel<<<grid_dim, block_dim, 0, stream>>>(
    input.get_const_base_ptr(),
    mean.get_const_base_ptr(),
    var.get_const_base_ptr(),
    scale.get_const_base_ptr(),
    bias.get_const_base_ptr(),
    output.get_base_ptr(),
    epsilon,
    shape,
    input_strides,
    output_strides);
}

}  // namespace

template <typename TensorType>
void batch_normalization(int num_dims,
                         int num_samples,
                         TensorType const& input,
                         TensorType const& mean,
                         TensorType const& var,
                         TensorType const& scale,
                         TensorType const& bias,
                         TensorType& output,
                         typename TensorType::data_type epsilon,
                         h2::gpu::DeviceStream stream)
{
  switch (num_dims)
  {
  case 4:
    batch_normalization<4, TensorType>(
      num_samples, input, mean, var, scale, bias, output, epsilon, stream);
    break;
  case 5:
    batch_normalization<5, TensorType>(
      num_samples, input, mean, var, scale, bias, output, epsilon, stream);
    break;
  }
}

#define INSTANTIATE_BATCH_NORMALIZATION(TYPE)                                  \
  template void batch_normalization<Tensor<TYPE>>(                             \
    int num_dims,                                                              \
    int num_samples,                                                           \
    const Tensor<TYPE>& input,                                                 \
    const Tensor<TYPE>& mean,                                                  \
    const Tensor<TYPE>& var,                                                   \
    const Tensor<TYPE>& scale,                                                 \
    const Tensor<TYPE>& bias,                                                  \
    Tensor<TYPE>& output,                                                      \
    TYPE epsilon,                                                              \
    h2::gpu::DeviceStream stream);
INSTANTIATE_BATCH_NORMALIZATION(float)
INSTANTIATE_BATCH_NORMALIZATION(double)
#undef INSTANTIATE_BATCH_NORMALIZATION

#ifdef DISTCONV_HAS_NVSHMEM
namespace
{

template <int ND,
          typename DataType,
          typename DataType2,
          typename DataTypeV,
          int BLOCK_SIZE>
__global__ void forward_all_kernel(DataTypeV const* __restrict__ input,
                                   DataType* __restrict__ running_mean,
                                   DataType* __restrict__ running_var,
                                   DataType const* __restrict__ scale,
                                   DataType const* __restrict__ bias,
                                   DataTypeV* __restrict__ output,
                                   DataType decay,
                                   DataType epsilon,
                                   int const sample_size,
                                   int const channel_size,
                                   int const spatial_size,
                                   int const spatial_real_size,
                                   size_t const num_per_sum,
                                   AllreduceNVSHMEMDevice<DataType2> ar)
{
  __shared__ DataType2 shared_stat[BLOCK_SIZE];
  int const tid = threadIdx.x;
  int const bid = blockIdx.x;
  auto const sample_offset = spatial_real_size * channel_size;

  index_t offset = spatial_real_size * bid;
  DataType2 stat = {DataType(0), DataType(0)};

  for (int s = 0; s < sample_size; ++s)
  {
    for (int i = tid; i < spatial_size; i += BLOCK_SIZE)
    {
      auto const x = input[offset + i];
      stat.x += util::sum(x);
      stat.y += util::sum(x * x);
    }
    offset += sample_offset;
  }

  shared_stat[tid] = stat;

  // Compute channel sum with shared memory reduction
#pragma unroll
  for (int stride = BLOCK_SIZE / 2; stride > 0; stride /= 2)
  {
    __syncthreads();
    if (tid < stride)
    {
      shared_stat[tid] += shared_stat[tid + stride];
    }
  }

  stat = shared_stat[0];

  // Output channel sum to global memory
  int const ch_idx = blockIdx.x;
  if (tid == 0)
  {
    // Assumes only one block per entry
    stat = ar.recursive_doubling_block(stat, 1);
    stat.x = stat.x / num_per_sum;
    stat.y = stat.y / num_per_sum;
    auto v = stat.y - stat.x * stat.x;
    v = max(v, DataType(0));
    v *= num_per_sum / (num_per_sum - DataType(1));
    stat.y = v;
    running_mean[ch_idx] =
      decay * running_mean[ch_idx] + (DataType(1) - decay) * stat.x;
    running_var[ch_idx] =
      decay * running_var[ch_idx] + (DataType(1) - decay) * stat.y;

    stat.y = rsqrt(stat.y + epsilon);
    shared_stat[0] = stat;
  }
  __syncthreads();
  stat = shared_stat[0];

  // fuse the batch_normalization kernel here
  auto const scale_ch = scale[ch_idx];
  auto const bias_ch = bias[ch_idx];

  offset = spatial_real_size * bid;

  for (int s = 0; s < sample_size; ++s)
  {
    for (int i = tid; i < spatial_size; i += BLOCK_SIZE)
    {
      auto idx = offset + i;
      auto const x = input[idx];
      auto xhat = (x - stat.x) * stat.y;
      auto y = xhat * scale_ch + bias_ch;
      output[idx] = y;
    }
    offset += sample_offset;
  }
}

template <int ND, typename Tensor>
void forward_all(Tensor const& input,
                 Tensor& mean,
                 Tensor& var,
                 Tensor& running_mean,
                 Tensor& running_var,
                 Tensor& scale,
                 Tensor& bias,
                 Tensor& output,
                 typename Tensor::data_type decay,
                 typename Tensor::data_type epsilon,
                 h2::gpu::DeviceStream stream,
                 AllreduceNVSHMEM<typename Tensor::data_type>& ar)
{
  using DataType = typename Tensor::data_type;
  using DataType2 = typename util::GetVectorType<DataType, 2>::type;

  auto const shape = input.get_local_shape();
  auto const real_shape = input.get_local_real_shape();
  int const num_samples = shape[get_sample_dim()];
  int const num_channels = shape[get_channel_dim()];

  int spatial_size = shape[0] * shape[1];
  int spatial_real_size = real_shape[0] * real_shape[1];
  if (ND >= 5)
  {
    spatial_size *= shape[2];
    spatial_real_size *= real_shape[2];
  }

  // Assumes halo can only be attached to the outermost spatial
  // dimension
  auto overlap = input.get_overlap();
  assert_eq(overlap[0], 0);
  if (ND >= 5)
  {
    assert_eq(overlap[1], 0);
  }

  constexpr int block_size = 1024;
  dim3 block_dim(block_size);
  dim3 grid_dim(num_channels);
  // CUDA grid dimension limitation
  assert_always(grid_dim.x < 65535);

  ar.recursive_doubling_block_setup(num_channels * 2, 1);

  auto num_per_sum = input.get_size() / input.get_shape()[-2];

  assert_always(input.get_local_size() > 0 && input.is_split_root());

  auto ar_dev = ar.template get_for_device<DataType2>();
  if (spatial_size % 4 == 0 && spatial_real_size % 4 == 0)
  {
    spatial_size /= 4;
    spatial_real_size /= 4;
    using DataTypeV = typename util::GetVectorType<DataType, 4>::type;
    forward_all_kernel<ND, DataType, DataType2, DataTypeV, block_size>
      <<<grid_dim, block_dim, 0, stream>>>(
        reinterpret_cast<DataTypeV const*>(input.get_const_base_ptr()),
        running_mean.get_base_ptr(),
        running_var.get_base_ptr(),
        scale.get_base_ptr(),
        bias.get_base_ptr(),
        reinterpret_cast<DataTypeV*>(output.get_base_ptr()),
        decay,
        epsilon,
        num_samples,
        num_channels,
        spatial_size,
        spatial_real_size,
        num_per_sum,
        ar_dev);
  }
  else
  {
    forward_all_kernel<ND, DataType, DataType2, DataType, block_size>
      <<<grid_dim, block_dim, 0, stream>>>(input.get_const_base_ptr(),
                                           running_mean.get_base_ptr(),
                                           running_var.get_base_ptr(),
                                           scale.get_base_ptr(),
                                           bias.get_base_ptr(),
                                           output.get_base_ptr(),
                                           decay,
                                           epsilon,
                                           num_samples,
                                           num_channels,
                                           spatial_size,
                                           spatial_real_size,
                                           num_per_sum,
                                           ar_dev);
  }
}

}  // namespace

template <typename Tensor>
void forward_all(int num_dims,
                 Tensor const& input,
                 Tensor& mean,
                 Tensor& var,
                 Tensor& running_mean,
                 Tensor& running_var,
                 Tensor& scale,
                 Tensor& bias,
                 Tensor& output,
                 typename Tensor::data_type decay,
                 typename Tensor::data_type epsilon,
                 h2::gpu::DeviceStream stream,
                 AllreduceNVSHMEM<typename Tensor::data_type>& ar)
{
  switch (num_dims)
  {
  case 4:
    forward_all<4, Tensor>(input,
                           mean,
                           var,
                           running_mean,
                           running_var,
                           scale,
                           bias,
                           output,
                           decay,
                           epsilon,
                           stream,
                           ar);
    break;
  case 5:
    forward_all<5, Tensor>(input,
                           mean,
                           var,
                           running_mean,
                           running_var,
                           scale,
                           bias,
                           output,
                           decay,
                           epsilon,
                           stream,
                           ar);
    break;
  }
}

#define INSTANTIATE_FORWARD(TYPE)                                              \
  template void forward_all<Tensor<TYPE>>(int num_dims,                        \
                                          const Tensor<TYPE>& input,           \
                                          Tensor<TYPE>& mean,                  \
                                          Tensor<TYPE>& var,                   \
                                          Tensor<TYPE>& running_mean,          \
                                          Tensor<TYPE>& running_var,           \
                                          Tensor<TYPE>& scale,                 \
                                          Tensor<TYPE>& bias,                  \
                                          Tensor<TYPE>& output,                \
                                          TYPE decay,                          \
                                          TYPE epsilon,                        \
                                          h2::gpu::DeviceStream stream,        \
                                          AllreduceNVSHMEM<TYPE>& ar);
INSTANTIATE_FORWARD(float)
INSTANTIATE_FORWARD(double)
#undef INSTANTIATE_FORWARD
#endif  // DISTCONV_HAS_NVSHMEM

namespace
{

template <int ND, typename DataType, int BLOCK_SIZE>
__global__ void backprop1_kernel(DataType const* __restrict__ input,
                                 DataType const* __restrict__ d_output,
                                 DataType const* __restrict__ global_mean,
                                 DataType const* __restrict__ global_var,
                                 DataType const* __restrict__ global_scale,
                                 DataType* __restrict__ global_dscale,
                                 DataType* __restrict__ global_dbias,
                                 DataType* __restrict__ global_dmean,
                                 DataType* __restrict__ global_dvar,
                                 DataType epsilon,
                                 tensor::Array<ND> shape,
                                 tensor::Array<ND> input_strides,
                                 tensor::Array<ND> d_output_strides)
{
  int const tid = threadIdx.x;
  index_t const gidx = threadIdx.x + blockIdx.x * blockDim.x;
  int const ch_idx = blockIdx.y;
  int const num_channels = shape[get_channel_dim()];
  int const num_samples = shape[get_sample_dim()];

  DataType const mean = global_mean[ch_idx];
  DataType const var = global_var[ch_idx];
  DataType const scale = global_scale[ch_idx];
  DataType const inv_stdev = rsqrt(var + epsilon);
  DataType const dvar_factor = inv_stdev * inv_stdev * inv_stdev / 2;

  DataType dscale = DataType(0);
  DataType dbias = DataType(0);
  DataType dmean = DataType(0);
  DataType dvar = DataType(0);

  index_t const channel_size = shape.get_size() / num_channels / num_samples;

  if (gidx < channel_size)
  {
    index_t offset = gidx;
    index_t input_offset = 0, d_output_offset = 0;
    for (int d = 0; d < ND - 2; ++d)
    {
      int idx = offset % shape[d];
      input_offset += idx * input_strides[d];
      d_output_offset += idx * d_output_strides[d];
      offset /= shape[d];
    }
    input_offset += ch_idx * input_strides[-2];
    d_output_offset += ch_idx * d_output_strides[-2];
    for (int sample_idx = 0; sample_idx < num_samples; ++sample_idx)
    {
      DataType const x = input[input_offset];
      DataType const xhat = (x - mean) * inv_stdev;
      DataType const dy = d_output[d_output_offset];
      dscale += dy * xhat;
      dbias += dy;
      DataType const dxhat = dy * scale;
      dmean += -dxhat * inv_stdev;
      dvar += -dxhat * (x - mean) * dvar_factor;

      input_offset += input_strides[-1];
      d_output_offset += d_output_strides[-1];
    }
  }

  using BlockReduce = cubns::BlockReduce<DataType, BLOCK_SIZE>;
  __shared__ typename BlockReduce::TempStorage temp_storage_scale;
  __shared__ typename BlockReduce::TempStorage temp_storage_bias;
  __shared__ typename BlockReduce::TempStorage temp_storage_mean;
  __shared__ typename BlockReduce::TempStorage temp_storage_var;
  dscale = BlockReduce(temp_storage_scale).Sum(dscale);
  dbias = BlockReduce(temp_storage_bias).Sum(dbias);
  dmean = BlockReduce(temp_storage_mean).Sum(dmean);
  dvar = BlockReduce(temp_storage_var).Sum(dvar);

  // Output channel sum to global memory
  if (tid == 0)
  {
    atomic_add(&global_dscale[ch_idx], dscale);
    atomic_add(&global_dbias[ch_idx], dbias);
    atomic_add(&global_dmean[ch_idx], dmean);
    atomic_add(&global_dvar[ch_idx], dvar);
  }
}

template <int ND, typename DataType, int BLOCK_SIZE, typename DataTypeV>
__global__ void backprop1_opt_kernel(DataTypeV const* __restrict__ input,
                                     DataTypeV const* __restrict__ d_output,
                                     DataType const* __restrict__ global_mean,
                                     DataType const* __restrict__ global_var,
                                     DataType const* __restrict__ global_scale,
                                     DataType* __restrict__ global_dscale,
                                     DataType* __restrict__ global_dbias,
                                     DataType* __restrict__ global_dmean,
                                     DataType* __restrict__ global_dvar,
                                     DataType epsilon,
                                     int const num_channels,
                                     int const num_samples,
                                     index_t const spatial_size,
                                     index_t const input_spatial_real_size,
                                     index_t const output_spatial_real_size)
{
  int const tid = threadIdx.x;
  index_t const idx = threadIdx.x + blockIdx.x * blockDim.x;
  int const ch_idx = blockIdx.y;
  auto const i_sample_offset = input_spatial_real_size * num_channels;
  auto const o_sample_offset = output_spatial_real_size * num_channels;

  auto const mean = global_mean[ch_idx];
  auto const var = global_var[ch_idx];
  auto const scale = global_scale[ch_idx];
  auto const inv_stdev = rsqrt(var + epsilon);
  auto const dvar_factor = inv_stdev * inv_stdev * inv_stdev / 2;

  DataType dscale = DataType(0);
  DataType dbias = DataType(0);
  DataType dmean = DataType(0);
  DataType dvar = DataType(0);

  index_t i_offset = input_spatial_real_size * ch_idx;
  index_t o_offset = output_spatial_real_size * ch_idx;

  for (int s = 0; s < num_samples; ++s)
  {
    for (auto i = idx; i < spatial_size; i += BLOCK_SIZE * gridDim.x)
    {
      auto const x = input[i_offset + i];
      auto const xhat = (x - mean) * inv_stdev;
      auto const dy = d_output[o_offset + i];
      dscale += util::sum(dy * xhat);
      dbias += util::sum(dy);
      auto const dxhat = dy * scale;
      dmean -= util::sum(dxhat * inv_stdev);
      dvar -= util::sum(dxhat * (x - mean) * dvar_factor);
    }
    i_offset += i_sample_offset;
    o_offset += o_sample_offset;
  }

  using BlockReduce = cubns::BlockReduce<DataType, BLOCK_SIZE>;
  __shared__ typename BlockReduce::TempStorage temp_storage_scale;
  __shared__ typename BlockReduce::TempStorage temp_storage_bias;
  __shared__ typename BlockReduce::TempStorage temp_storage_mean;
  __shared__ typename BlockReduce::TempStorage temp_storage_var;
  dscale = BlockReduce(temp_storage_scale).Sum(dscale);
  dbias = BlockReduce(temp_storage_bias).Sum(dbias);
  dmean = BlockReduce(temp_storage_mean).Sum(dmean);
  dvar = BlockReduce(temp_storage_var).Sum(dvar);

  // Output channel sum to global memory
  if (tid == 0)
  {
    atomic_add(&global_dscale[ch_idx], dscale);
    atomic_add(&global_dbias[ch_idx], dbias);
    atomic_add(&global_dmean[ch_idx], dmean);
    atomic_add(&global_dvar[ch_idx], dvar);
  }
}

template <int ND, typename TensorType>
void backprop1_opt(int num_samples,
                   TensorType const& input,
                   TensorType const& d_output,
                   TensorType const& mean,
                   TensorType const& var,
                   TensorType const& scale,
                   TensorType& scale_gradient,
                   TensorType& bias_gradient,
                   TensorType& mean_gradient,
                   TensorType& var_gradient,
                   typename TensorType::data_type epsilon,
                   h2::gpu::DeviceStream stream)
{
  using DataType = typename TensorType::data_type;
  int const num_channels = input.get_local_shape()[get_channel_dim()];
  constexpr int block_size = 256;
  dim3 block_dim(block_size);
  constexpr index_t thread_work_size = 8;
  constexpr auto block_work_size = block_size * thread_work_size;
  index_t spatial_size = input.get_local_size() / num_channels / num_samples;
  index_t i_spatial_real_size =
    input.get_local_real_size() / num_channels / num_samples;
  index_t o_spatial_real_size =
    d_output.get_local_real_size() / num_channels / num_samples;
  // halo size must be also divisible by a vector width for an
  // alignment requirement
  if (spatial_size % 4 == 0
      && ((i_spatial_real_size - spatial_size) / 2) % 4 == 0
      && ((o_spatial_real_size - spatial_size) / 2) % 4 == 0)
  {
    using DataTypeV = typename util::GetVectorType<DataType, 4>::type;
    spatial_size /= 4;
    i_spatial_real_size /= 4;
    o_spatial_real_size /= 4;
    auto num_blocks_per_channel = util::ceil(spatial_size, block_work_size);
    dim3 grid_dim(num_blocks_per_channel, num_channels);
    backprop1_opt_kernel<ND, DataType, block_size, DataTypeV>
      <<<grid_dim, block_dim, 0, stream>>>(
        reinterpret_cast<DataTypeV const*>(input.get_const_base_ptr()),
        reinterpret_cast<DataTypeV const*>(d_output.get_const_base_ptr()),
        mean.get_const_base_ptr(),
        var.get_const_base_ptr(),
        scale.get_const_base_ptr(),
        scale_gradient.get_base_ptr(),
        bias_gradient.get_base_ptr(),
        mean_gradient.get_base_ptr(),
        var_gradient.get_base_ptr(),
        epsilon,
        num_channels,
        num_samples,
        spatial_size,
        i_spatial_real_size,
        o_spatial_real_size);
  }
  else
  {
    using DataTypeV = DataType;
    auto num_blocks_per_channel = util::ceil(spatial_size, block_work_size);
    dim3 grid_dim(num_blocks_per_channel, num_channels);
    backprop1_opt_kernel<ND, DataType, block_size, DataTypeV>
      <<<grid_dim, block_dim, 0, stream>>>(input.get_const_base_ptr(),
                                           d_output.get_const_base_ptr(),
                                           mean.get_const_base_ptr(),
                                           var.get_const_base_ptr(),
                                           scale.get_const_base_ptr(),
                                           scale_gradient.get_base_ptr(),
                                           bias_gradient.get_base_ptr(),
                                           mean_gradient.get_base_ptr(),
                                           var_gradient.get_base_ptr(),
                                           epsilon,
                                           num_channels,
                                           num_samples,
                                           spatial_size,
                                           i_spatial_real_size,
                                           o_spatial_real_size);
  }
}

template <int ND, typename TensorType>
void backprop1(int num_samples,
               TensorType const& input,
               TensorType const& d_output,
               TensorType const& mean,
               TensorType const& var,
               TensorType const& scale,
               TensorType& scale_gradient,
               TensorType& bias_gradient,
               TensorType& mean_gradient,
               TensorType& var_gradient,
               typename TensorType::data_type epsilon,
               h2::gpu::DeviceStream stream)
{
  using DataType = typename TensorType::data_type;
  h2::gpu::mem_zero(scale_gradient.get_buffer(),
                    scale_gradient.get_local_pitched_size(),
                    stream);
  h2::gpu::mem_zero(
    bias_gradient.get_buffer(), bias_gradient.get_local_pitched_size(), stream);
  h2::gpu::mem_zero(
    mean_gradient.get_buffer(), mean_gradient.get_local_pitched_size(), stream);
  h2::gpu::mem_zero(
    var_gradient.get_buffer(), var_gradient.get_local_pitched_size(), stream);

  if (input.get_local_size() == 0 || !input.is_split_root())
  {
    return;
  }

  std::vector<IndexVector> overlaps = {input.get_overlap(),
                                       d_output.get_overlap()};
  bool opt_eligible = true;
  for (auto overlap : overlaps)
  {
    for (int i = 0; i < ND - 3; ++i)
    {
      if (overlap[i] != 0)
      {
        opt_eligible = false;
        break;
      }
    }
  }
  if (std::getenv("DISTCONV_DISABLE_BN_OPT"))
  {
    util::MPIRootPrintStreamInfo() << "Disable BN optimization";
    opt_eligible = false;
  }
  if (opt_eligible)
  {
    backprop1_opt<ND, TensorType>(num_samples,
                                  input,
                                  d_output,
                                  mean,
                                  var,
                                  scale,
                                  scale_gradient,
                                  bias_gradient,
                                  mean_gradient,
                                  var_gradient,
                                  epsilon,
                                  stream);
    return;
  }

  auto const input_strides = input.get_strides();
  auto const d_output_strides = d_output.get_strides();
  int const num_channels = input.get_local_shape()[get_channel_dim()];
  // CUDA grid dimension limitation
  assert_always(num_channels < 65535);
  constexpr int block_size = 256;
  dim3 block_dim(block_size);
  auto shape = input.get_local_shape();
  shape[get_sample_dim()] = num_samples;
  index_t channel_size = input.get_local_size() / num_channels / num_samples;
  dim3 grid_dim((channel_size + block_size - 1) / block_size, num_channels);
  backprop1_kernel<ND, DataType, block_size>
    <<<grid_dim, block_dim, 0, stream>>>(input.get_const_base_ptr(),
                                         d_output.get_const_base_ptr(),
                                         mean.get_const_base_ptr(),
                                         var.get_const_base_ptr(),
                                         scale.get_const_base_ptr(),
                                         scale_gradient.get_base_ptr(),
                                         bias_gradient.get_base_ptr(),
                                         mean_gradient.get_base_ptr(),
                                         var_gradient.get_base_ptr(),
                                         epsilon,
                                         shape,
                                         input_strides,
                                         d_output_strides);
}

}  // namespace

template <typename TensorType>
void backprop1(int num_dims,
               int num_samples,
               TensorType const& input,
               TensorType const& d_output,
               TensorType const& mean,
               TensorType const& var,
               TensorType const& scale,
               TensorType& scale_gradient,
               TensorType& bias_gradient,
               TensorType& mean_gradient,
               TensorType& var_gradient,
               typename TensorType::data_type epsilon,
               h2::gpu::DeviceStream stream)
{
  switch (num_dims)
  {
  case 4:
    backprop1<4, TensorType>(num_samples,
                             input,
                             d_output,
                             mean,
                             var,
                             scale,
                             scale_gradient,
                             bias_gradient,
                             mean_gradient,
                             var_gradient,
                             epsilon,
                             stream);
    break;
  case 5:
    backprop1<5, TensorType>(num_samples,
                             input,
                             d_output,
                             mean,
                             var,
                             scale,
                             scale_gradient,
                             bias_gradient,
                             mean_gradient,
                             var_gradient,
                             epsilon,
                             stream);
    break;
  }
}

#define INSTANTIATE_BACKPROP1(TYPE)                                            \
  template void backprop1<Tensor<TYPE>>(int num_dims,                          \
                                        int num_samples,                       \
                                        const Tensor<TYPE>& input,             \
                                        const Tensor<TYPE>& d_output,          \
                                        const Tensor<TYPE>& mean,              \
                                        const Tensor<TYPE>& var,               \
                                        const Tensor<TYPE>& scale,             \
                                        Tensor<TYPE>& scale_gradient,          \
                                        Tensor<TYPE>& bias_gradient,           \
                                        Tensor<TYPE>& mean_gradient,           \
                                        Tensor<TYPE>& var_gradient,            \
                                        TYPE epsilon,                          \
                                        h2::gpu::DeviceStream stream);
INSTANTIATE_BACKPROP1(float)
INSTANTIATE_BACKPROP1(double)
#undef INSTANTIATE_BACKPROP1

namespace
{

template <int ND, typename DataType>
__global__ void backprop2_kernel(
  DataType const* input,  // no __restrict__ so input can be reused for d_input
                          // as a memory optimization
  DataType const* __restrict__ d_output,
  DataType const* __restrict__ global_mean,
  DataType const* __restrict__ global_var,
  DataType const* __restrict__ global_scale,
  DataType const* __restrict__ global_dmean,
  DataType const* __restrict__ global_dvar,
  DataType* d_input,  // no __restrict__ so input can be reused for d_input as
                      // a memory optimization
  DataType epsilon,
  index_t num_per_sum,
  tensor::Array<ND> shape,
  tensor::Array<ND> input_strides,
  tensor::Array<ND> d_output_strides,
  tensor::Array<ND> d_input_strides)
{
  index_t const gidx = threadIdx.x + blockIdx.x * blockDim.x;
  int const ch_idx = blockIdx.y;
  int const num_channels = shape[get_channel_dim()];
  int const num_samples = shape[-1];

  DataType const mean = global_mean[ch_idx];
  DataType const var = global_var[ch_idx];
  DataType const scale = global_scale[ch_idx];
  DataType const dmean = global_dmean[ch_idx];
  DataType const dvar = global_dvar[ch_idx];

  DataType const inv_stdev = rsqrt(var + epsilon);
  DataType const dmean_term = dmean / num_per_sum;
  DataType const dvar_term = dvar * 2 / (num_per_sum - 1);

  index_t const channel_size = shape.get_size() / num_channels / num_samples;

  if (gidx < channel_size)
  {
    index_t offset = gidx;
    index_t input_offset = 0, d_output_offset = 0, d_input_offset = 0;
    for (int d = 0; d < ND - 2; ++d)
    {
      int idx = offset % shape[d];
      input_offset += idx * input_strides[d];
      d_output_offset += idx * d_output_strides[d];
      d_input_offset += idx * d_input_strides[d];
      offset /= shape[d];
    }
    input_offset += ch_idx * input_strides[-2];
    d_output_offset += ch_idx * d_output_strides[-2];
    d_input_offset += ch_idx * d_input_strides[-2];
    for (int s = 0; s < num_samples; ++s)
    {
      DataType const x = input[input_offset];
      DataType const dy = d_output[d_output_offset];
      DataType const dxhat = dy * scale;
      DataType dx = dxhat * inv_stdev;
      dx += dmean_term;
      dx += dvar_term * (x - mean);
      d_input[d_input_offset] = dx;

      input_offset += input_strides[-1];
      d_output_offset += d_output_strides[-1];
      d_input_offset += d_input_strides[-1];
    }
  }
}

template <int ND, typename DataType, typename DataTypeV>
__global__ void backprop2_opt_kernel(
  DataTypeV const* input,  // no __restrict__ so input can be reused for
                           // d_input as a memory optimization
  DataTypeV const* __restrict__ d_output,
  DataType const* __restrict__ global_mean,
  DataType const* __restrict__ global_var,
  DataType const* __restrict__ global_scale,
  DataType const* __restrict__ global_dmean,
  DataType const* __restrict__ global_dvar,
  DataTypeV* d_input,  // no __restrict__ so input can be reused for d_input as
                       // a memory optimization
  DataType epsilon,
  index_t num_per_sum,
  index_t spatial_size,
  int num_channels)
{
  auto const ch_idx = blockIdx.y;
  auto const sample_idx = blockIdx.z;
  auto const mean = global_mean[ch_idx];
  auto const var = global_var[ch_idx];
  auto const scale = global_scale[ch_idx];
  auto const dmean = global_dmean[ch_idx];
  auto const dvar = global_dvar[ch_idx];
  auto const inv_stdev = rsqrt(var + epsilon);
  auto const dmean_term = dmean / num_per_sum;
  auto const dvar_term = dvar * 2 / (num_per_sum - 1);

  auto const num_threads_per_channel = blockDim.x * gridDim.x;

  auto block_offset = (ch_idx + sample_idx * num_channels) * spatial_size;
  input += block_offset;
  d_output += block_offset;
  d_input += block_offset;

  for (index_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < spatial_size;
       idx += num_threads_per_channel)
  {
    auto const x = input[idx];
    auto const dy = d_output[idx];
    auto const dxhat = dy * scale;
    auto dx = dxhat * inv_stdev;
    dx = dx + dmean_term;
    dx = dx + (x - mean) * dvar_term;
    d_input[idx] = dx;
  }
}

template <int ND, typename TensorType>
void backprop2_opt(index_t num_samples,
                   index_t num_per_sum,
                   TensorType const& input,
                   TensorType const& d_output,
                   TensorType const& mean,
                   TensorType const& var,
                   TensorType const& scale,
                   TensorType const& mean_gradient,
                   TensorType const& var_gradient,
                   TensorType& d_input,
                   typename TensorType::data_type epsilon,
                   h2::gpu::DeviceStream stream)
{
  using DataType = typename TensorType::data_type;
  // local tensors can be empty
  if (input.get_local_size() == 0)
    return;
  assert_eq(num_samples, (int) input.get_local_shape()[get_sample_dim()]);
  int const num_channels = input.get_local_shape()[get_channel_dim()];
  constexpr int block_size = 256;
  dim3 block_dim(block_size);
  index_t channel_size = input.get_local_size() / num_channels / num_samples;
  constexpr index_t thread_work_size = 8;
  constexpr auto block_work_size = block_size * thread_work_size;
  if (channel_size % 4 == 0)
  {
    channel_size /= 4;
    auto num_blocks_per_channel = util::ceil(channel_size, block_work_size);
    dim3 grid_dim(num_blocks_per_channel, num_channels, num_samples);
    using DataTypeV = typename util::GetVectorType<DataType, 4>::type;
    backprop2_opt_kernel<ND, DataType, DataTypeV>
      <<<grid_dim, block_dim, 0, stream>>>(
        reinterpret_cast<DataTypeV const*>(input.get_const_buffer()),
        reinterpret_cast<DataTypeV const*>(d_output.get_const_buffer()),
        mean.get_const_base_ptr(),
        var.get_const_base_ptr(),
        scale.get_const_base_ptr(),
        mean_gradient.get_const_base_ptr(),
        var_gradient.get_const_base_ptr(),
        reinterpret_cast<DataTypeV*>(d_input.get_buffer()),
        epsilon,
        num_per_sum,
        channel_size,
        num_channels);
  }
  else
  {
    auto num_blocks_per_channel = util::ceil(channel_size, block_work_size);
    dim3 grid_dim(num_blocks_per_channel, num_channels, num_samples);
    backprop2_opt_kernel<ND, DataType, DataType>
      <<<grid_dim, block_dim, 0, stream>>>(input.get_const_buffer(),
                                           d_output.get_const_buffer(),
                                           mean.get_const_base_ptr(),
                                           var.get_const_base_ptr(),
                                           scale.get_const_base_ptr(),
                                           mean_gradient.get_const_base_ptr(),
                                           var_gradient.get_const_base_ptr(),
                                           d_input.get_buffer(),
                                           epsilon,
                                           num_per_sum,
                                           channel_size,
                                           num_channels);
  }
}

template <int ND, typename TensorType>
void backprop2(index_t num_samples,
               index_t num_per_sum,
               TensorType const& input,
               TensorType const& d_output,
               TensorType const& mean,
               TensorType const& var,
               TensorType const& scale,
               TensorType const& mean_gradient,
               TensorType const& var_gradient,
               TensorType& d_input,
               typename TensorType::data_type epsilon,
               h2::gpu::DeviceStream stream)
{
  using DataType = typename TensorType::data_type;

  if (input.get_local_real_shape() == d_output.get_local_real_shape()
      && input.get_local_real_shape() == d_input.get_local_real_shape()
      && input.get_overlap() == 0 && d_output.get_overlap() == 0
      && d_input.get_overlap() == 0)
  {
    if (std::getenv("DISTCONV_DISABLE_BN_OPT"))
    {
      util::MPIRootPrintStreamInfo() << "Disable BN optimization";
    }
    else
    {
      backprop2_opt<ND, TensorType>(num_samples,
                                    num_per_sum,
                                    input,
                                    d_output,
                                    mean,
                                    var,
                                    scale,
                                    mean_gradient,
                                    var_gradient,
                                    d_input,
                                    epsilon,
                                    stream);
      return;
    }
  }

  if (d_input.get_local_size() == 0)
    return;
  int const num_channels = input.get_local_shape()[get_channel_dim()];
  constexpr int block_size = 256;
  dim3 block_dim(block_size);
  index_t channel_size = input.get_local_size() / num_channels / num_samples;
  dim3 grid_dim((channel_size + block_size - 1) / block_size, num_channels);
  auto input_strides = input.get_strides();
  auto d_output_strides = d_output.get_strides();
  auto d_input_strides = d_input.get_strides();
  auto shape = input.get_local_shape();
  shape[get_sample_dim()] = num_samples;
  // CUDA grid dimension limitation
  assert_always(num_channels < 65535);
  backprop2_kernel<ND, DataType>
    <<<grid_dim, block_dim, 0, stream>>>(input.get_const_base_ptr(),
                                         d_output.get_const_base_ptr(),
                                         mean.get_const_base_ptr(),
                                         var.get_const_base_ptr(),
                                         scale.get_const_base_ptr(),
                                         mean_gradient.get_const_base_ptr(),
                                         var_gradient.get_const_base_ptr(),
                                         d_input.get_base_ptr(),
                                         epsilon,
                                         num_per_sum,
                                         shape,
                                         input_strides,
                                         d_output_strides,
                                         d_input_strides);
}

}  // namespace

template <typename TensorType>
void backprop2(int num_dims,
               index_t num_samples,
               index_t num_per_sum,
               TensorType const& input,
               TensorType const& d_output,
               TensorType const& mean,
               TensorType const& var,
               TensorType const& scale,
               TensorType const& mean_gradient,
               TensorType const& var_gradient,
               TensorType& d_input,
               typename TensorType::data_type epsilon,
               h2::gpu::DeviceStream stream)
{
  switch (num_dims)
  {
  case 4:
    backprop2<4, TensorType>(num_samples,
                             num_per_sum,
                             input,
                             d_output,
                             mean,
                             var,
                             scale,
                             mean_gradient,
                             var_gradient,
                             d_input,
                             epsilon,
                             stream);
    break;
  case 5:
    backprop2<5, TensorType>(num_samples,
                             num_per_sum,
                             input,
                             d_output,
                             mean,
                             var,
                             scale,
                             mean_gradient,
                             var_gradient,
                             d_input,
                             epsilon,
                             stream);
    break;
  }
}

#define INSTANTIATE_BACKPROP2(TYPE)                                            \
  template void backprop2<Tensor<TYPE>>(int num_dims,                          \
                                        index_t num_samples,                   \
                                        index_t num_per_sum,                   \
                                        const Tensor<TYPE>& input,             \
                                        const Tensor<TYPE>& d_output,          \
                                        const Tensor<TYPE>& mean,              \
                                        const Tensor<TYPE>& var,               \
                                        const Tensor<TYPE>& scale,             \
                                        const Tensor<TYPE>& mean_gradient,     \
                                        const Tensor<TYPE>& var_gradient,      \
                                        Tensor<TYPE>& d_input,                 \
                                        TYPE epsilon,                          \
                                        h2::gpu::DeviceStream stream);
INSTANTIATE_BACKPROP2(float)
INSTANTIATE_BACKPROP2(double)
#undef INSTANTIATE_BACKPROP2

}  // namespace batchnorm
}  // namespace distconv
