#include "hip/hip_runtime.h"
#include "distconv/dnn_backend/pooling.hpp"
#include "distconv/runtime_gpu.hpp"
#include "distconv/util/util_mpi.hpp"

namespace
{

// using namespace distconv;
namespace dc = distconv;
namespace tensor = dc::tensor;
namespace util = dc::util;
using index_t = dc::index_t;

template <int ND>
using Array = tensor::Array<ND>;

template <typename DataType>
using Tensor =
  tensor::Tensor<DataType, tensor::LocaleMPI, tensor::CUDAAllocator>;

template <int ND, typename DataType>
__global__ void bp_accumulate_sum_kernel(DataType* tensor,
                                         const Array<ND> tensor_shape,
                                         const Array<ND> dst_offset,
                                         const Array<ND> src_offset,
                                         const Array<ND> region)
{
  index_t idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= region.get_size())
    return;
  Array<ND> offset;
  for (int i = 0; i < ND; ++i)
  {
    offset[i] = idx % region[i];
    idx = idx / region[i];
  }
  DataType src = tensor[tensor::get_offset(src_offset + offset, tensor_shape)];
  DataType& dst = tensor[tensor::get_offset(dst_offset + offset, tensor_shape)];
  dst += src;
#if 0
  if (blockIdx.x == 0 && threadIdx.x < 32) {
    printf("DST: %d\n",
           (int)tensor::get_offset(dst_offset + offset, tensor_shape));
  }
#endif
}

template <int ND, typename DataType>
void bp_accumulate_sum_nd(Tensor<DataType>& tensor,
                          const dc::IndexVector& dst,
                          const dc::IndexVector& src,
                          const dc::tensor::Shape& shape)
{
  auto size = shape.get_size();
  const int bsize = 256;
  int gsize = (size + bsize - 1) / bsize;
  util::MPIPrintStreamDebug()
    << "Accumulating " << src << " to " << dst << " of sub tensor with shape "
    << shape << " of " << tensor.get_local_pitched_shape() << "\n";

#if 0
  {
    DISTCONV_CHECK_CUDA(hipDeviceSynchronize());
    DataType *h = nullptr;
    size_t s = tensor.get_local_real_size() * sizeof(DataType);
    hipHostMalloc(&h, s);
    DISTCONV_CHECK_CUDA(hipMemcpy(h, tensor.get_buffer(), s,
                                   hipMemcpyDeviceToHost));
    std::ofstream out;
    out.open("before.out",
             std::ios::out | std::ios::trunc | std::ios::binary);
    out.write((char *)h, s);
    out.close();
    DISTCONV_CHECK_CUDA(hipHostFree(h));
  }
#endif

  bp_accumulate_sum_kernel<ND><<<gsize, bsize>>>(
    tensor.get_buffer(), tensor.get_local_pitched_shape(), dst, src, shape);
#if 0
  {
    DataType *h = nullptr;
    size_t s = tensor.get_local_real_size() * sizeof(DataType);
    hipHostMalloc(&h, s);
    DISTCONV_CHECK_CUDA(hipDeviceSynchronize());
    DISTCONV_CHECK_CUDA(hipMemcpy(h, tensor.get_buffer(), s,
                                   hipMemcpyDeviceToHost));
    std::ofstream out;
    out.open("after.out",
             std::ios::out | std::ios::trunc | std::ios::binary);
    out.write((char *)h, s);
    out.close();
    DISTCONV_CHECK_CUDA(hipHostFree(h));
  }
#endif
}

}  // namespace

namespace distconv
{

template <typename DataType>
void Pooling<BackendDNNLib, DataType>::bp_accumulate_sum(
  Tensor<DataType>& tensor,
  IndexVector const& dst,
  IndexVector const& src,
  tensor::Shape const& shape)
{
  switch (m_num_dims)
  {
  case 4: bp_accumulate_sum_nd<4, DataType>(tensor, dst, src, shape); break;
  case 5: bp_accumulate_sum_nd<5, DataType>(tensor, dst, src, shape); break;
  }
  return;
}

#define INSTANTIATE_BP_ACCUMULATE_SUM(TYPE)                                    \
  template void Pooling<BackendDNNLib, TYPE>::bp_accumulate_sum(               \
    Tensor<TYPE>& tensor,                                                      \
    const IndexVector& dst,                                                    \
    const IndexVector& src,                                                    \
    const tensor::Shape& shape)
INSTANTIATE_BP_ACCUMULATE_SUM(float);
INSTANTIATE_BP_ACCUMULATE_SUM(double);
#undef INSTANTIATE_BP_ACCUMULATE_SUM

}  // namespace distconv
