#include "hip/hip_runtime.h"
#include "distconv/dnn_backend/mean_squared_error.hpp"
#include "distconv/runtime_gpu.hpp"
#include "distconv/tensor/algorithms_cuda.hpp"
#include "distconv/util/util_gpu.hpp"
#include "distconv/util/util_mpi.hpp"

#include <limits>

#if H2_HAS_CUDA
#include <cub/block/block_reduce.cuh>
namespace cubns = cub;
#elif H2_HAS_ROCM
#include <hipcub/block/block_reduce.hpp>
namespace cubns = hipcub;
#endif

using distconv::tensor::LocaleMPI;
using distconv::tensor::CUDAAllocator;

template <typename DataType>
using TensorCUDA = distconv::tensor::Tensor<DataType, LocaleMPI, CUDAAllocator>;
using MeanSquaredErrorCUDNN =
    distconv::MeanSquaredError<distconv::BackendDNNLib>;

namespace distconv {
namespace mean_squared_error {

/*
  - gridDim.y == number of samples
  - Each sample is taken care by gridDim.x blocks
*/
template <typename DataType, int BLOCK_SIZE>
__global__ void fp_local(const DataType * __restrict__ prediction,
                         const DataType * __restrict__ ground_truth,
                         DataType * __restrict__ y,
                         const index_t sample_size,
                         const index_t sample_spatial_size,
                         const index_t sample_channel_size,
                         int thread_work_size) {
  const int tid = threadIdx.x;
  const int sample_idx = blockIdx.y;

  prediction += sample_idx * sample_size;
  ground_truth += sample_idx * sample_size;

  index_t offset = tid + blockIdx.x * BLOCK_SIZE;
  const int offset_stride = BLOCK_SIZE * gridDim.x;
  const index_t offset_limit = min(
      sample_size, offset + offset_stride * thread_work_size);

  auto psum = DataType(0.);
  for (; offset < offset_limit; offset += offset_stride) {
    const DataType x = prediction[offset];
    const DataType xhat = ground_truth[offset];
    const DataType err = x - xhat;
    psum += err * err;
  }

  using BlockReduce = cubns::BlockReduce<DataType, BLOCK_SIZE>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  psum = BlockReduce(temp_storage).Sum(psum) / sample_size;

  if (tid == 0) {
    atomic_add(&y[sample_idx], psum);
  }
}


/*
  - gridDim.y == number of samples
  - Each sample is taken care by gridDim.x blocks
*/
template <typename DataType, int BLOCK_SIZE>
__global__ void bp_local(const DataType * __restrict__ x_pred,
                         const DataType * __restrict__ x_truth,
                         const DataType * __restrict__ dy,
                         DataType * __restrict__ dx_pred,
                         DataType * __restrict__ dx_truth,
                         const index_t sample_size,
                         const index_t sample_spatial_size,
                         const index_t sample_channel_size,
                         int thread_work_size) {
  const int tid = threadIdx.x;
  const int sample_idx = blockIdx.y;

  x_pred += sample_idx * sample_size;
  dx_pred += sample_idx * sample_size;
  x_truth += sample_idx * sample_size;
  dx_truth += sample_idx * sample_size;

  index_t offset = tid + blockIdx.x * BLOCK_SIZE;
  const int offset_stride = BLOCK_SIZE * gridDim.x;
  const index_t offset_limit = min(
      sample_size, offset + offset_stride * thread_work_size);

  const auto dy_sample = dy[sample_idx];
  const DataType scale = static_cast<DataType>(DataType(2) / sample_size);
  for (; offset < offset_limit; offset += offset_stride) {
    const DataType x = x_pred[offset];
    const DataType xhat = x_truth[offset];
    const DataType err = x - xhat;
    dx_pred[offset] = scale * err * dy_sample;
    dx_truth[offset] = - scale * err * dy_sample;
  }
}

} // namespace mean_squared_error

template <typename Tensor>
int MeanSquaredErrorCUDNN::forward(const Tensor &x_pred, const Tensor &x_truth,
                                   Tensor &y) {
  using DataType = typename Tensor::data_type;
  util::MPIPrintStreamDebug()
      << "Mean squared error FP: " << x_pred << ", "
      << x_truth << ", " << y;

  constexpr int block_size = 256;
  constexpr int thread_work_size = 8;

  // Assumes no halo for simplicity
  assert_eq(x_pred.get_local_size(), x_pred.get_local_real_size());
  assert_eq(x_truth.get_local_size(), x_truth.get_local_real_size());

  const auto num_samples = x_pred.get_local_shape()[-1];

  if (num_samples == 0) return 0;

  y.zero(m_be.get_stream());

  if (x_pred.get_local_size() > 0) {
    auto sample_size = x_pred.get_local_size() / num_samples;
    auto num_blocks_per_sample = util::ceil(
        sample_size, (index_t)block_size * thread_work_size);

    dim3 bdim(block_size);
    dim3 gdim(num_blocks_per_sample, num_samples);

    const auto sample_channel_size = x_pred.get_local_shape()[x_pred.get_num_spatial_dims()];
    const auto sample_spatial_size = sample_size / sample_channel_size;
    assert_eq(sample_channel_size*sample_spatial_size, sample_size);

    mean_squared_error::fp_local<DataType, block_size>
        <<<gdim, bdim, 0, m_be.get_stream()>>>(
            x_pred.get_const_buffer(), x_truth.get_const_buffer(),
            y.get_buffer(), sample_size, sample_spatial_size,
            sample_channel_size, thread_work_size);
  }

  if (m_num_procs_per_sample > 1) {
    Al::Allreduce<Al::NCCLBackend, DataType>(
        y.get_buffer(), num_samples,
        Al::ReductionOperator::sum, *m_al.get());
  }

  return 0;
}

template <typename Tensor>
int MeanSquaredErrorCUDNN::backward(const Tensor &x_pred, const Tensor &x_truth,
                                    Tensor &dy, Tensor &dx_pred,
                                    Tensor &dx_truth) {
  using DataType = typename Tensor::data_type;
  util::MPIPrintStreamDebug()
      << "Mean squared error BP: " << dy << ", " << dx_pred << ", " << dx_truth;

  if (m_num_procs_per_sample > 1) {
    const auto num_samples = x_pred.get_local_shape()[-1];
    Al::Bcast<Al::NCCLBackend, DataType>(
        dy.get_buffer(), num_samples, 0,
        *m_al.get());
  }

  constexpr int block_size = 256;
  constexpr int thread_work_size = 8;

  // Assumes no halo for simplicity
  assert_eq(dx_pred.get_local_size(), dx_pred.get_local_real_size());
  assert_eq(dx_truth.get_local_size(), dx_truth.get_local_real_size());

  if (x_pred.get_local_size() == 0) return 0;

  auto num_samples = x_pred.get_local_shape()[-1];
  auto sample_size = x_pred.get_local_size() / num_samples;
  auto num_blocks_per_sample = util::ceil(
      sample_size, (index_t)block_size * thread_work_size);

  dim3 bdim(block_size);
  dim3 gdim(num_blocks_per_sample, num_samples);

  const auto sample_channel_size = x_pred.get_local_shape()[x_pred.get_num_spatial_dims()];
  const auto sample_spatial_size = sample_size / sample_channel_size;
  assert_eq(sample_channel_size*sample_spatial_size, sample_size);

  mean_squared_error::bp_local<DataType, block_size>
      <<<gdim, bdim, 0, m_be.get_stream()>>>(
          x_pred.get_const_buffer(), x_truth.get_const_buffer(),
          dy.get_const_buffer(),
          dx_pred.get_buffer(), dx_truth.get_buffer(),
          sample_size, sample_spatial_size, sample_channel_size,
          thread_work_size);
  return 0;
}

#define PROTO(T)                                                        \
  template int MeanSquaredErrorCUDNN::forward<TensorCUDA<T>>(           \
      const TensorCUDA<T> &x_pred, const TensorCUDA<T> &x_truth,        \
      TensorCUDA<T> &y);                                                \
  template int MeanSquaredErrorCUDNN::backward<TensorCUDA<T>>(          \
      const TensorCUDA<T> &x_pred, const TensorCUDA<T> &x_truth,        \
      TensorCUDA<T> &dy, TensorCUDA<T> &dx_pred,                        \
      TensorCUDA<T> &dx_truth);

PROTO(float)
PROTO(double)
#undef PROTO

} // namespace distconv
