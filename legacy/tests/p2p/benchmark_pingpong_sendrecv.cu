#include "hip/hip_runtime.h"
#include <cassert>
#include <chrono>
#include <cstdlib>
#include <iostream>
#include <sstream>
#include <thread>

#include "hip/hip_runtime_api.h"
#include "p2p/logging.hpp"
#include "p2p/p2p.hpp"
#include "p2p/util.hpp"
#include "p2p/util_cuda.hpp"
#include "test_util.hpp"
#include "test_util_cuda.hpp"

int test_bandwidth(const int pid,
                   p2p::P2P& p2p,
                   const size_t min_size,
                   const size_t max_size,
                   const int iter)
{
  void* buf1;
  P2P_CHECK_CUDA_ALWAYS(hipMalloc(&buf1, max_size));
  hipStream_t st;
  P2P_CHECK_CUDA_ALWAYS(hipStreamCreate(&st));
  const int peer = (pid % 2) == 0 ? pid + 1 : pid - 1;
  p2p::P2P::connection_type conn;
  p2p.get_connections(&peer, &conn, 1);
  hipEvent_t ev1, ev2;
  P2P_CHECK_CUDA_ALWAYS(hipEventCreate(&ev1));
  P2P_CHECK_CUDA_ALWAYS(hipEventCreate(&ev2));
  const int skip = 5;

  p2p::logging::MPIRootPrintStreamInfo()
    << "Min size: " << min_size << ", max size: " << max_size << "\n";

  for (size_t size = min_size; size <= max_size; size *= 2)
  {
    p2p::logging::MPIRootPrintStreamDebug() << "Testing " << size << "\n";
    P2P_CHECK_CUDA_ALWAYS(hipStreamSynchronize(st));
    MPI_Barrier(MPI_COMM_WORLD);

    if (pid == 0)
    {
      spin_device(st, 1);
    }

    for (int i = 0; i < iter + skip; ++i)
    {
      if (i == skip)
      {
        P2P_CHECK_CUDA(hipEventRecord(ev1, st));
      }
      if (pid == 0)
      {
        conn->send(buf1, size, st);
        conn->recv(buf1, size, st);
      }
      else
      {
        conn->recv(buf1, size, st);
        conn->send(buf1, size, st);
      }
    }
    P2P_CHECK_CUDA(hipEventRecord(ev2, st));
    P2P_CHECK_CUDA_ALWAYS(hipStreamSynchronize(st));

    float elapsed;
    P2P_CHECK_CUDA(hipEventElapsedTime(&elapsed, ev1, ev2));
    elapsed = elapsed / 2 * 1000 / iter;
    if (pid == 0)
    {
      std::stringstream ss;
      ss << size << " " << elapsed << "\n";
      std::cout << ss.str();
    }
  }
  MPI_Barrier(MPI_COMM_WORLD);
  p2p::logging::MPIPrintStreamInfo() << __FUNCTION__ << " done\n";
  p2p.disconnect_all();
  return 0;
}

int main(int argc, char* argv[])
{
  size_t min_size = 1;
  size_t max_size = 1 << 24;
  int iter = 10;

  assert(argc <= 4);

  if (argc == 2)
  {
    min_size = std::atol(argv[1]);
    max_size = min_size;
  }
  if (argc >= 3)
  {
    min_size = std::atol(argv[1]);
    max_size = std::atol(argv[2]);
  }
  if (argc >= 4)
  {
    iter = std::atoi(argv[3]);
  }

  int local_rank = get_local_rank();
  std::cerr << "local rank: " << local_rank << "\n";

  P2P_CHECK_CUDA_ALWAYS(hipSetDevice(local_rank));
  std::cerr << "cuda devise set: " << local_rank << "\n";

  int mpi_thread_level;
  MPI_Init_thread(&argc, &argv, MPI_THREAD_MULTIPLE, &mpi_thread_level);
  switch (mpi_thread_level)
  {
  case MPI_THREAD_MULTIPLE:
    std::cout << "Supported thread level: MPI_THREAD_MULTIPLE\n";
    break;
  case MPI_THREAD_SINGLE:
    std::cout << "Supported thread level: MPI_THREAD_SINGLE\n";
    break;
  case MPI_THREAD_SERIALIZED:
    std::cout << "Supported thread level: MPI_THREAD_SERIALIZED\n";
    break;
  case MPI_THREAD_FUNNELED:
    std::cout << "Supported thread level: MPI_THREAD_FUNNELED\n";
    break;
  }
  int pid;
  int np;
  MPI_Comm_rank(MPI_COMM_WORLD, &pid);
  MPI_Comm_size(MPI_COMM_WORLD, &np);
  P2P_ASSERT_ALWAYS(np == 2);

  p2p::P2P p2p(MPI_COMM_WORLD);

  // p2p.enable_nvtx();

  TEST_RUN(test_bandwidth(pid, p2p, min_size, max_size, iter));

  p2p.disconnect_all();

  MPI_Finalize();
  return 0;
}
