#include "hip/hip_runtime.h"
#include "distconv/runtime_gpu.hpp"
#include "distconv/tensor/tensor.hpp"
#include "distconv/tensor/tensor_cuda.hpp"
#include "distconv/tensor/tensor_mpi.hpp"
#include "distconv/tensor/tensor_mpi_cuda.hpp"
#include "distconv/util/util_gpu.hpp"
#include "distconv/util/util_mpi.hpp"

#include <assert.h>

#include <iostream>
#include <vector>

#include "test_tensor.hpp"

using namespace distconv;
using namespace distconv::tensor;
using namespace distconv::util;

template <>
inline LocaleMPI get_locale()
{
  LocaleMPI loc(MPI_COMM_WORLD);
  return loc;
}

__global__ void init_tensor(int* buf,
                            Array<3> local_shape,
                            Array<3> halo,
                            index_t pitch,
                            Array<3> global_shape,
                            Array<3> global_index_base)
{
  Array<3> local_real_shape = local_shape + halo * 2;
  for (index_t k = blockIdx.x; k < local_shape[2]; k += gridDim.x)
  {
    for (index_t j = 0; j < local_shape[1]; ++j)
    {
      for (index_t i = threadIdx.x; i < local_shape[0]; i += blockDim.x)
      {
        Array<3> local_idx = {i, j, k};
        size_t local_offset =
          get_offset(local_idx + halo, local_real_shape, pitch);
        Array<3> global_idx = global_index_base + local_idx;
        size_t global_offset = get_offset(global_idx, global_shape);
        buf[local_offset] = global_offset;
      }
    }
  }
}

__global__ void check_tensor(const int* buf,
                             Array<3> local_shape,
                             Array<3> halo,
                             index_t pitch,
                             Array<3> global_shape,
                             Array<3> global_index_base,
                             int* error_counter)
{
  Array<3> local_real_shape = local_shape + halo * 2;
  for (index_t k = blockIdx.x; k < local_shape[2]; k += gridDim.x)
  {
    for (index_t j = 0; j < local_shape[1]; ++j)
    {
      for (index_t i = threadIdx.x; i < local_shape[0]; i += blockDim.x)
      {
        Array<3> local_idx = {i, j, k};
        size_t local_offset =
          get_offset(local_idx + halo, local_real_shape, pitch);
        Array<3> global_idx = global_index_base + local_idx;
        int global_offset = get_offset(global_idx, global_shape);
        int stored = buf[local_offset];
        if (stored != global_offset)
        {
          atomicAdd(error_counter, 1);
          printf(
            "Error at (%lu, %lu, %lu)@(%lu, %lu, %lu); ref: %d, stored: %d\n",
            global_idx[0],
            global_idx[1],
            global_idx[2],
            i,
            j,
            k,
            global_offset,
            stored);
        }
      }
    }
  }
}

template <typename TensorType>
inline int test_data_access_mpi_cuda(const Shape& shape,
                                     const Distribution& dist)
{
  using LocaleType = typename TensorType::locale_type;
  LocaleType loc = get_locale<LocaleType>();
  TensorType t = get_tensor<TensorType>(shape, loc, dist);
  MPIRootPrintStreamDebug() << "Shape: " << t.get_shape();
  MPIRootPrintStreamDebug() << "Distribution: " << t.get_distribution();
  MPIPrintStreamDebug() << "Local real shape: " << t.get_local_real_shape();

  assert0(t.allocate());

  // Array<3> local_shape = t.get_local_shape();
  index_t base_offset = t.get_local_offset();
  int* buf = t.get_buffer();
  assert_always(buf != nullptr);
  size_t pitch = t.get_pitch();
  util::MPIPrintStreamDebug()
    << "Base offset: " << base_offset
    << ", global offset: " << t.get_global_index() << ", pitch: " << pitch;

  init_tensor<<<4, 4>>>(buf,
                        t.get_local_shape(),
                        dist.get_overlap(),
                        t.get_pitch(),
                        t.get_shape(),
                        t.get_global_index());
  h2::gpu::sync();

  int error_counter = 0;
  int* error_counter_d;
  GPU_MALLOC(&error_counter_d, sizeof(int));
  h2::gpu::mem_copy(error_counter_d, &error_counter);
  check_tensor<<<1, 1>>>(buf,
                         t.get_local_shape(),
                         dist.get_overlap(),
                         t.get_pitch(),
                         t.get_shape(),
                         t.get_global_index(),
                         error_counter_d);

  h2::gpu::mem_copy(&error_counter, error_counter_d);
  assert0(error_counter);

  return 0;
}

template <typename TensorType>
int test_view_raw_ptr(const Shape& shape, const Distribution& dist)
{
  auto loc = get_locale<typename TensorType::locale_type>();
  auto t = get_tensor<TensorType>(shape, loc, dist);
  assert0(t.allocate());

  index_t base_offset = t.get_local_offset();
  int* buf = t.get_buffer();
  assert_always(buf);
  init_tensor<<<4, 4>>>(buf,
                        t.get_local_shape(),
                        dist.get_overlap(),
                        t.get_pitch(),
                        t.get_shape(),
                        t.get_global_index());
  h2::gpu::sync();
  using ConstTensorType = Tensor<typename TensorType::data_type,
                                 typename TensorType::locale_type,
                                 typename TensorType::allocator_type>;
  auto const_tensor_view = get_tensor<ConstTensorType>(shape, loc, dist);
  View(const_tensor_view, (const int*) buf);
  assert_always(const_tensor_view.get_const_buffer() == buf);
  int error_counter = 0;
  int* error_counter_d;
  GPU_MALLOC(&error_counter_d, sizeof(int));
  h2::gpu::mem_copy(error_counter_d, &error_counter);
  check_tensor<<<1, 1>>>(const_tensor_view.get_const_buffer(),
                         const_tensor_view.get_local_shape(),
                         dist.get_overlap(),
                         const_tensor_view.get_pitch(),
                         const_tensor_view.get_shape(),
                         const_tensor_view.get_global_index(),
                         error_counter_d);
  h2::gpu::mem_copy(&error_counter, error_counter_d);
  assert0(error_counter);
  return 0;
}

template <int ND, typename DataType>
__global__ void check_clear_halo(const DataType* buf,
                                 Array<ND> local_shape,
                                 int dim,
                                 int halo,
                                 DataType default_value,
                                 int* error_counter)
{
  const int tid = threadIdx.x;
  const int num_threads = blockDim.x;
  Array<ND> idx;
  idx[1] = blockIdx.x;
  idx[2] = blockIdx.y;
  if (ND == 4)
  {
    idx[3] = blockIdx.z;
  }

  for (int x = tid; x < local_shape[0]; x += num_threads)
  {
    idx[0] = x;
    int offset = get_offset(idx, local_shape);
    DataType v = buf[offset];
    if (idx[dim] < halo || idx[dim] >= local_shape[dim] - halo)
    {
      if (v != 0)
      {
        atomicAdd(error_counter, 1);
      }
    }
    else
    {
      if (v != default_value)
      {
        atomicAdd(error_counter, 1);
      }
    }
  }
}

template <int ND, typename TensorType>
int test_clear_halo(const Shape& shape, const Distribution& dist)
{
  const int num_dims = shape.num_dims();
  using DataType = typename TensorType::data_type;
  using LocaleType = typename TensorType::locale_type;
  LocaleType loc = get_locale<LocaleType>();
  TensorType t = get_tensor<TensorType>(shape, loc, dist);
  const auto local_real_shape = t.get_local_real_shape();
  util::MPIPrintStreamDebug()
    << "Shape: " << t.get_shape() << ", local real shape: " << local_real_shape
    << ", distribution: " << t.get_distribution();

  int error_counter = 0;
  int* error_counter_d;
  GPU_MALLOC(&error_counter_d, sizeof(int));
  h2::gpu::mem_copy(error_counter_d, &error_counter);

  assert0(t.allocate());
  auto* buf = t.get_buffer();
  std::vector<DataType> hvec;
  hvec.reserve(t.get_local_real_size());
  auto* h = hvec.data();
  DataType default_value = 1;
  for (size_t i = 0; i < t.get_local_real_size(); ++i)
  {
    h[i] = default_value;
  }
  for (int i = 0; i < num_dims; ++i)
  {
    h2::gpu::mem_copy(buf, h, t.get_local_real_size());
    t.clear_halo(i);
    dim3 gsize(local_real_shape[1], local_real_shape[2]);
    if (num_dims == 4)
    {
      gsize.z = local_real_shape[3];
    }
    check_clear_halo<ND, DataType><<<gsize, 128>>>(
      buf, local_real_shape, i, dist.get_overlap(i), 1, error_counter_d);
    h2::gpu::mem_copy(&error_counter, error_counter_d);
    if (error_counter != 0)
    {
      util::MPIPrintStreamError() << error_counter << " errors at dimension ";
      h2::gpu::mem_copy(h, buf, t.get_local_real_size());
      std::ofstream out;
      std::ostringstream file_path;
      file_path << "clear_halo_test_" << loc.get_rank();
      out.open(file_path.str(), std::ios::out | std::ios::trunc);
      for (size_t i = 0; i < t.get_local_real_size(); ++i)
      {
        out << h[i] << "\n";
      }
      out.close();
      return -1;
    }
#if 0
    // FIXME: Whenever this gets un-"#if 0"-ed, we should use a vector.
    DataType *result_h = new DataType[t.get_local_real_size()];
    h2::gpu::mem_copy(result_h, buf, t.get_local_real_size());
    std::ofstream out;
    std::ostringstream file_path;
    file_path << "clear_halo_test_" << i << "_" << loc.get_rank();
    out.open(file_path.str(), std::ios::out | std::ios::trunc);
    for (size_t i = 0; i < t.get_local_real_size(); ++i) {
      out << result_h[i] << "\n";
    }
    out.close();
#endif
  }

  return 0;
}

/*
  Usage: mpirun -np N ./test_tensor_mpi_cuda, where N must be >= 8 and
  divisible by 8.
 */
int main(int argc, char* argv[])
{
  h2::gpu::set_gpu(util::choose_gpu());
  MPI_Init(&argc, &argv);
  int pid;
  int np;
  MPI_Comm_rank(MPI_COMM_WORLD, &pid);
  MPI_Comm_size(MPI_COMM_WORLD, &np);

  MPIPrintStreamInfo() << "Using device " << h2::gpu::current_gpu();

  constexpr int ND = 3;
  using DataType = int;

  using TensorMPI = Tensor<DataType, LocaleMPI, CUDAAllocator>;
  auto dist3 =
    Distribution::make_overlapped_distribution({2, 2, np / 4}, {1, 1, 0});
  auto dist4 =
    Distribution::make_overlapped_distribution({2, 2, 2, np / 8}, {1, 1, 0, 0});
  assert_always((np % 8) == 0 && (np >= 8));
  // Distribution<3> dist({1, 1, np}, {1, 1, 0});

  assert0(test_alloc<TensorMPI>(Shape({2, 2, 2}), dist3));
  MPIRootPrintStreamInfo() << "test_alloc success";

  assert0(test_data_access_mpi_cuda<TensorMPI>(Shape({2, 2, 2}), dist3));
  MPIRootPrintStreamInfo() << "test_data_access_mpi_cuda success";

  // Doesn't work with Spectrum-MPI
#if 0
  assert0(test_data_access_mpi_cuda<Tensor<DataType, LocaleMPI,
          CUDAPitchedAllocator>>(Shape({32, 32, 4}), dist3));
  MPIRootPrintStreamInfo() << "test_data_access_mpi_cuda with pitched memory success\n";
#endif

  assert0(test_view_raw_ptr<TensorMPI>(Shape({32, 32, 4}), dist3));

  MPIRootPrintStreamInfo() << "test_view_raw_ptr success";

  assert0(test_clear_halo<ND, TensorMPI>(Shape({32, 31, 4}), dist3));
  MPIRootPrintStreamInfo() << "test_clear_halo success";

  assert0(test_clear_halo<4, TensorMPI>(Shape({32, 31, 4, 8}), dist4));
  MPIRootPrintStreamInfo() << "test_clear_halo with 4D tensor success";

  MPI_Finalize();

  DISTCONV_CHECK_GPU(GPU_DEVICE_RESET());
  return 0;
}
